#include "hip/hip_runtime.h"
#include <finite_element/finite_element_method.h>
#include <Eigen/Dense>
#include <Eigen/Geometry>
#include <finite_element/finite_element_extra_constitution.h>
#include <finite_element/finite_element_constitution.h>
#include <uipc/builtin/attribute_name.h>
#include <uipc/geometry/simplicial_complex.h>
#include <uipc/common/map.h>
#include <uipc/common/zip.h>
#include <finite_element/fem_utils.h>
#include <uipc/common/algorithm/run_length_encode.h>
#include <uipc/common/json_eigen.h>
#include <muda/ext/eigen/inverse.h>
#include <ranges>
#include <sim_engine.h>
#include <utils/offset_count_collection.h>

// kinetic
#include <finite_element/finite_element_kinetic.h>
// constitutions
#include <finite_element/fem_3d_constitution.h>
#include <finite_element/codim_2d_constitution.h>
#include <finite_element/codim_1d_constitution.h>
#include <finite_element/codim_0d_constitution.h>
#include <uipc/builtin/constitution_type.h>
// diff parm reporters
#include <finite_element/finite_element_diff_parm_reporter.h>
#include <finite_element/finite_element_constitution_diff_parm_reporter.h>
#include <finite_element/finite_element_extra_constitution_diff_parm_reporter.h>


namespace uipc::backend
{
template <>
class backend::SimSystemCreator<cuda::FiniteElementMethod>
{
  public:
    static U<cuda::FiniteElementMethod> create(SimEngine& engine)
    {
        auto  scene = dynamic_cast<cuda::SimEngine&>(engine).world().scene();
        auto& types = scene.constitution_tabular().types();
        if(types.find(std::string{builtin::FiniteElement}) == types.end())
        {
            return nullptr;
        }
        return uipc::make_unique<cuda::FiniteElementMethod>(engine);
    }
};
}  // namespace uipc::backend


bool operator<(const uipc::backend::cuda::FiniteElementMethod::DimUID& a,
               const uipc::backend::cuda::FiniteElementMethod::DimUID& b)
{
    return a.dim < b.dim || (a.dim == b.dim && a.uid < b.uid);
}

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(FiniteElementMethod);

void FiniteElementMethod::do_build()
{
    const auto& scene = world().scene();

    m_impl.default_gravity = scene.info()["gravity"].get<Vector3>();

    m_impl.global_vertex_manager = &require<GlobalVertexManager>();

    // Register the action to write the scene
    on_write_scene([this] { m_impl.write_scene(world()); });
}

IndexT FiniteElementMethod::dof_offset(SizeT frame) const noexcept
{
    return m_impl.dof_offset(frame);
}

IndexT FiniteElementMethod::dof_count(SizeT frame) const noexcept
{
    return m_impl.dof_count(frame);
}

void FiniteElementMethod::add_constitution(FiniteElementConstitution* constitution)
{
    check_state(SimEngineState::BuildSystems, "add_constitution()");
    m_impl.constitutions.register_subsystem(*constitution);
}

void FiniteElementMethod::add_constitution(FiniteElementExtraConstitution* constitution)
{
    check_state(SimEngineState::BuildSystems, "add_constitution()");
    m_impl.extra_constitutions.register_subsystem(*constitution);
}

void FiniteElementMethod::add_constitution(FiniteElementKinetic* constitution)
{
    check_state(SimEngineState::BuildSystems, "add_constitution()");
    m_impl.kinetic.register_subsystem(*constitution);
}

void FiniteElementMethod::add_reporter(FiniteElementConstitutionDiffParmReporter* reporter)
{
    //check_state(SimEngineState::BuildSystems, "add_reporter()");
    //m_impl.constitution_diff_parm_reporters.register_subsystem(*reporter);
}

void FiniteElementMethod::add_reporter(FiniteElementExtraConstitutionDiffParmReporter* reporter)
{
    //check_state(SimEngineState::BuildSystems, "add_reporter()");
    //m_impl.extra_constitution_diff_parm_reporters.register_subsystem(*reporter);
}

void FiniteElementMethod::add_kinetic_reporter(FiniteElementDiffParmReporter* reporter)
{
    //check_state(SimEngineState::BuildSystems, "add_reporter()");
    //m_impl.kinetic_diff_parm_reporter.register_subsystem(*reporter);
}

void FiniteElementMethod::init()
{
    m_impl.init(world());
}

bool FiniteElementMethod::do_dump(DumpInfo& info)
{
    return m_impl.dump(info);
}

bool FiniteElementMethod::do_try_recover(RecoverInfo& info)
{
    return m_impl.try_recover(info);
}

void FiniteElementMethod::do_apply_recover(RecoverInfo& info)
{
    m_impl.apply_recover(info);
}

void FiniteElementMethod::do_clear_recover(RecoverInfo& info)
{
    m_impl.clear_recover(info);
}

void FiniteElementMethod::Impl::init(WorldVisitor& world)
{
    _init_dof_info();

    // 1) setup base constitution infos
    _classify_base_constitutions();
    _build_geo_infos(world);
    _build_base_constitution_infos();
    _build_on_host(world);
    _build_on_device();

    // 2) Init event
    _init_base_constitution();
    _init_extra_constitutions();
    _init_energy_producers();
    _init_diff_reporters();
}

void FiniteElementMethod::Impl::_init_dof_info()
{
    frame_to_dof_count.reserve(1024);
    frame_to_dof_offset.reserve(1024);
    // frame 0 is not used
    frame_to_dof_offset.push_back(0);
    frame_to_dof_count.push_back(0);
}

void FiniteElementMethod::Impl::_classify_base_constitutions()
{
    auto constitution_view = constitutions.view();

    // 1) sort the constitutions by (dim, uid)
    std::sort(constitution_view.begin(),
              constitution_view.end(),
              [](const FiniteElementConstitution* a, const FiniteElementConstitution* b)
              {
                  auto   uida = a->uid();
                  auto   uidb = b->uid();
                  auto   dima = a->dim();
                  auto   dimb = b->dim();
                  DimUID uid_dim_a{dima, uida};
                  DimUID uid_dim_b{dimb, uidb};
                  return uid_dim_a < uid_dim_b;
              });

    for(auto&& [i, c] : enumerate(constitution_view))
        c->m_index = i;

    // 2) classify the constitutions
    codim_0d_constitutions.reserve(constitution_view.size());
    codim_1d_constitutions.reserve(constitution_view.size());
    codim_2d_constitutions.reserve(constitution_view.size());
    fem_3d_constitutions.reserve(constitution_view.size());

    for(auto&& constitution : constitution_view)
    {
        auto dim = constitution->dim();
        switch(dim)
        {
            case 0: {
                auto derived = dynamic_cast<Codim0DConstitution*>(constitution);
                UIPC_ASSERT(derived, "The constitution is not a Codim0DConstitution, its dim = {}", dim);
                derived->m_index_in_dim = codim_0d_constitutions.size();
                codim_0d_constitutions.push_back(derived);
                codim_0d_uid_to_index.insert({derived->uid(), derived->m_index_in_dim});
            }
            break;
            case 1: {
                auto derived = dynamic_cast<Codim1DConstitution*>(constitution);
                UIPC_ASSERT(derived, "The constitution is not a Codim1DConstitution, its dim = {}", dim);
                derived->m_index_in_dim = codim_1d_constitutions.size();
                codim_1d_constitutions.push_back(derived);
                codim_1d_uid_to_index.insert({derived->uid(), derived->m_index_in_dim});
            }
            break;
            case 2: {
                auto derived = dynamic_cast<Codim2DConstitution*>(constitution);
                UIPC_ASSERT(derived, "The constitution is not a Codim2DConstitution, its dim = {}", dim);
                derived->m_index_in_dim = codim_2d_constitutions.size();
                codim_2d_constitutions.push_back(derived);
                codim_2d_uid_to_index.insert({derived->uid(), derived->m_index_in_dim});
            }
            break;
            case 3: {
                auto derived = dynamic_cast<FEM3DConstitution*>(constitution);
                UIPC_ASSERT(derived, "The constitution is not a FEM3DConstitution, its dim = {}", dim);
                derived->m_index_in_dim = fem_3d_constitutions.size();
                fem_3d_constitutions.push_back(derived);
                fem_3d_uid_to_index.insert({derived->uid(), derived->m_index_in_dim});
            }
            break;
            default:
                break;
        }
    }
}

void FiniteElementMethod::Impl::_init_diff_reporters()
{
    //if(kinetic_diff_parm_reporter)
    //{
    //    kinetic_diff_parm_reporter.view()->init();
    //}

    //auto constitution_diff_parm_reporter_view = constitution_diff_parm_reporters.view();
    //auto extra_constitution_diff_parm_reporter_view =
    //    extra_constitution_diff_parm_reporters.view();

    //// 1. Connect the reporter to the related constitution
    //for(auto& cdpr : constitution_diff_parm_reporter_view)
    //{
    //    cdpr->connect();
    //}

    //for(auto& ecdpr : extra_constitution_diff_parm_reporter_view)
    //{
    //    ecdpr->connect();
    //}

    //// 2. Init the reporter
    //for(auto& cdpr : constitution_diff_parm_reporter_view)
    //{
    //    cdpr->init();
    //}

    //for(auto& ecdpr : extra_constitution_diff_parm_reporter_view)
    //{
    //    ecdpr->init();
    //}
}

void FiniteElementMethod::Impl::_build_geo_infos(WorldVisitor& world)
{
    set<U64> filter_uids;

    for(auto&& filter : constitutions.view())
        filter_uids.insert(filter->uid());

    // 1) find all the finite element constitutions
    auto geo_slots = world.scene().geometries();
    geo_infos.reserve(geo_slots.size());

    for(auto&& [i, geo_slot] : enumerate(geo_slots))
    {
        auto& geo  = geo_slot->geometry();
        auto  cuid = geo.meta().find<U64>(builtin::constitution_uid);
        if(cuid)
        {
            auto uid = cuid->view()[0];
            if(filter_uids.find(uid) != filter_uids.end())  // if exists
            {
                auto* sc = geo.as<geometry::SimplicialComplex>();
                UIPC_ASSERT(sc,
                            "The geometry is not a simplicial complex (it's {}). Why can it happen?",
                            geo.type());

                GeoInfo info;
                info.geo_slot_index = i;
                info.vertex_count   = sc->vertices().size();
                info.dim_uid.dim    = sc->dim();
                info.dim_uid.uid    = uid;

                switch(sc->dim())
                {
                    case 0:
                        info.primitive_count = sc->vertices().size();
                        break;
                    case 1:
                        info.primitive_count = sc->edges().size();
                        break;
                    case 2:
                        info.primitive_count = sc->triangles().size();
                        break;
                    case 3:
                        info.primitive_count = sc->tetrahedra().size();
                        break;
                    default:
                        break;
                }

                geo_infos.push_back(info);
            }
        }
    }

    // 2) sort geometry by (dim, uid)
    std::sort(geo_infos.begin(),
              geo_infos.end(),
              [](const GeoInfo& a, const GeoInfo& b)
              { return a.dim_uid < b.dim_uid; });


    // 3) setup vertex offsets and primitive offsets
    // + 1 for total count
    {
        OffsetCountCollection<IndexT> vertex_offsets_counts;
        vertex_offsets_counts.resize(geo_infos.size());
        OffsetCountCollection<IndexT> primitive_offsets_counts;
        primitive_offsets_counts.resize(geo_infos.size());

        span<IndexT> vertex_counts    = vertex_offsets_counts.counts();
        span<IndexT> primitive_counts = primitive_offsets_counts.counts();

        std::transform(geo_infos.begin(),
                       geo_infos.end(),
                       vertex_counts.begin(),
                       [](const GeoInfo& info) { return info.vertex_count; });

        std::transform(geo_infos.begin(),
                       geo_infos.end(),
                       primitive_counts.begin(),
                       [](const GeoInfo& info) { return info.primitive_count; });

        vertex_offsets_counts.scan();

        // we don't calculate the primitive offset here
        // the primitive offset is related to the dimension
        // the primitive offset in every dim starts from 0

        span<const IndexT> vertex_offsets = vertex_offsets_counts.offsets();

        for(auto&& [i, info] : enumerate(geo_infos))
        {
            info.vertex_offset = vertex_offsets[i];
        }

        h_positions.resize(vertex_offsets_counts.total_count());
    }


    // 4) setup dim infos
    {
        std::array<SizeT, 4> dim_geo_counts;
        std::array<SizeT, 4> dim_geo_offsets;
        dim_geo_counts.fill(0);


        vector<SizeT> offsets;
        offsets.reserve(dim_geo_counts.size());
        vector<SizeT> counts;
        counts.reserve(dim_geo_counts.size());

        // encode the dimension
        encode_offset_count(geo_infos.begin(),
                            geo_infos.end(),
                            std::back_inserter(offsets),
                            std::back_inserter(counts),
                            [](const GeoInfo& current, const GeoInfo& value)
                            { return current.dim_uid.dim == value.dim_uid.dim; });

        for(auto&& [offset, count] : zip(offsets, counts))
        {
            auto& info                       = geo_infos[offset];
            dim_geo_counts[info.dim_uid.dim] = count;
        }

        std::exclusive_scan(
            dim_geo_counts.begin(), dim_geo_counts.end(), dim_geo_offsets.begin(), 0);

        for(auto&& [i, dim_info] : enumerate(dim_infos))
        {
            dim_info.geo_info_offset = dim_geo_offsets[i];
            dim_info.geo_info_count  = dim_geo_counts[i];
        }
    }


    // 4) setup dim_info vertex and primitive
    vector<SizeT> dim_primitive_counts(dim_infos.size(), 0);
    vector<SizeT> dim_vertex_counts(dim_infos.size(), 0);
    vector<SizeT> dim_vertex_offsets(dim_infos.size(), 0);

    for(auto&& [i, dim_info] : enumerate(dim_infos))
    {
        auto it = std::find_if(geo_infos.begin(),
                               geo_infos.end(),
                               [i](const GeoInfo& info)
                               { return info.dim_uid.dim == i; });

        if(it == geo_infos.end())
            continue;

        OffsetCountCollection<IndexT> primitive_offsets_counts;
        primitive_offsets_counts.resize(dim_info.geo_info_count);

        span<IndexT>   primitive_counts = primitive_offsets_counts.counts();
        vector<IndexT> vertex_counts(dim_info.geo_info_count);

        auto geo_span =
            span{geo_infos}.subspan(dim_info.geo_info_offset, dim_info.geo_info_count);

        std::ranges::transform(geo_span,
                               primitive_counts.begin(),
                               [](const GeoInfo& info)
                               { return info.primitive_count; });

        // exclusive scan the primitive counts
        primitive_offsets_counts.scan();

        span<const IndexT> primitive_offsets = primitive_offsets_counts.offsets();

        for(auto&& [j, info] : enumerate(geo_span))
        {
            info.primitive_offset = primitive_offsets[j];
        }

        dim_primitive_counts[i] = primitive_offsets_counts.total_count();

        UIPC_ASSERT(geo_span.size() == vertex_counts.size(),
                    "Size mismatching in geo_span({}) and vertex_counts({}), why can it happen?",
                    geo_span.size(),
                    vertex_counts.size());

        std::ranges::transform(geo_span,
                               vertex_counts.begin(),
                               [](const GeoInfo& info)
                               { return info.vertex_count; });


        dim_vertex_counts[i] =
            std::accumulate(vertex_counts.begin(), vertex_counts.end(), 0);
    }

    std::exclusive_scan(dim_vertex_counts.begin(),
                        dim_vertex_counts.end(),
                        dim_vertex_offsets.begin(),
                        0);

    for(auto&& [i, dim_info] : enumerate(dim_infos))
    {
        dim_info.vertex_count     = dim_vertex_counts[i];
        dim_info.vertex_offset    = dim_vertex_offsets[i];
        dim_info.primitive_offset = 0;  // always 0
        dim_info.primitive_count  = dim_primitive_counts[i];
    }


    h_codim_0ds.resize(dim_infos[0].primitive_count);
    h_codim_1ds.resize(dim_infos[1].primitive_count);
    h_codim_2ds.resize(dim_infos[2].primitive_count);
    h_tets.resize(dim_infos[3].primitive_count);
}

void FiniteElementMethod::Impl::_build_base_constitution_infos()
{
    auto build_infos = [&]<std::derived_from<FiniteElementConstitution> ConstitutionT>(
                           vector<ConstitutionInfo>& infos,
                           span<ConstitutionT*>      constitutions,
                           IndexT                    dim,
                           unordered_map<U64, SizeT> uid_to_index)
    {
        infos.resize(constitutions.size());
        vector<SizeT> vertex_counts(infos.size(), 0);
        vector<SizeT> primitive_counts(infos.size(), 0);
        vector<SizeT> geometry_counts(infos.size(), 0);

        const auto& dim_info = dim_infos[dim];

        auto geo_info_subspan =
            span{geo_infos}.subspan(dim_info.geo_info_offset, dim_info.geo_info_count);


        for(auto&& geo_info : geo_info_subspan)
        {
            auto index = uid_to_index[geo_info.dim_uid.uid];
            geometry_counts[index]++;
            vertex_counts[index] += geo_info.vertex_count;
            primitive_counts[index] += geo_info.primitive_count;
        }

        vector<SizeT> vertex_offsets(infos.size(), 0);
        vector<SizeT> primitive_offsets(infos.size(), 0);
        vector<SizeT> geometry_offsets(infos.size(), 0);

        SizeT dim_geo_offset    = dim_info.geo_info_offset;
        SizeT dim_vertex_offset = 0;

        if(geo_infos.size() > dim_geo_offset)
        {
            const auto& begin_geo         = geo_infos[dim_geo_offset];
            SizeT       dim_vertex_offset = begin_geo.vertex_offset;
        }

        std::exclusive_scan(vertex_counts.begin(),
                            vertex_counts.end(),
                            vertex_offsets.begin(),
                            dim_vertex_offset);

        std::exclusive_scan(primitive_counts.begin(),
                            primitive_counts.end(),
                            primitive_offsets.begin(),
                            0);

        std::exclusive_scan(geometry_counts.begin(),
                            geometry_counts.end(),
                            geometry_offsets.begin(),
                            dim_geo_offset);

        for(auto&& [i, info] : enumerate(infos))
        {
            info.vertex_count     = vertex_counts[i];
            info.vertex_offset    = vertex_offsets[i];
            info.primitive_count  = primitive_counts[i];
            info.primitive_offset = primitive_offsets[i];
            info.geo_info_count   = geometry_counts[i];
            info.geo_info_offset  = geometry_offsets[i];
        }
        return 0;
    };


    build_infos(codim_0d_constitution_infos, span{codim_0d_constitutions}, 0, codim_0d_uid_to_index);
    build_infos(codim_1d_constitution_infos, span{codim_1d_constitutions}, 1, codim_1d_uid_to_index);
    build_infos(codim_2d_constitution_infos, span{codim_2d_constitutions}, 2, codim_2d_uid_to_index);
    build_infos(fem_3d_constitution_infos, span{fem_3d_constitutions}, 3, fem_3d_uid_to_index);
}

void FiniteElementMethod::Impl::_build_on_host(WorldVisitor& world)
{
    auto geo_slots      = world.scene().geometries();
    auto rest_geo_slots = world.scene().rest_geometries();

    // 1. Vertex Attributes
    {
        h_gravities.resize(h_positions.size(), default_gravity);
        h_rest_positions.resize(h_positions.size());
        h_velocities.resize(h_positions.size(), Vector3::Zero());  // fill 0 for default
        h_thicknesses.resize(h_positions.size(), 0);  // fill 0 for default
        h_dimensions.resize(h_positions.size(), 3);   // fill 3(D) for default
        h_masses.resize(h_positions.size());
        h_vertex_contact_element_ids.resize(h_positions.size(), 0);  // fill 0 for default
        h_vertex_is_fixed.resize(h_positions.size(), 0);  // fill 0 for default, default non-fixed
        h_vertex_is_dynamic.resize(h_positions.size(), 1);  // fill 1 for default, default dynamic
        h_vertex_body_id.resize(h_positions.size(), -1);  // fill -1 for default, invalid body id


        for(auto&& [i, info] : enumerate(geo_infos))
        {
            auto& geo_slot      = geo_slots[info.geo_slot_index];
            auto& rest_geo_slot = rest_geo_slots[info.geo_slot_index];
            auto& geo           = geo_slot->geometry();
            auto& rest_geo      = rest_geo_slot->geometry();
            auto* sc            = geo.as<geometry::SimplicialComplex>();
            UIPC_ASSERT(sc,
                        "The geometry is not a simplicial complex (it's {}). Why can it happen?",
                        geo.type());
            auto* rest_sc = rest_geo.as<geometry::SimplicialComplex>();
            UIPC_ASSERT(rest_sc,
                        "The geometry is not a simplicial complex (it's {}). Why can it happen?",
                        rest_geo.type());

            // 1) setup primitives
            switch(sc->dim())
            {
                case 0: {
                    auto dst_codim_0d_span =
                        span{h_codim_0ds}.subspan(info.primitive_offset, info.primitive_count);
                    std::iota(dst_codim_0d_span.begin(), dst_codim_0d_span.end(), info.vertex_offset);
                }
                break;
                case 1: {
                    auto dst_codim_1d_span =
                        span{h_codim_1ds}.subspan(info.primitive_offset, info.primitive_count);

                    auto edge_view = sc->edges().topo().view();
                    UIPC_ASSERT(edge_view.size() == dst_codim_1d_span.size(),
                                "edge size mismatching");

                    std::transform(edge_view.begin(),
                                   edge_view.end(),
                                   dst_codim_1d_span.begin(),
                                   [&](const Vector2i& edge) -> Vector2i
                                   { return edge.array() + info.vertex_offset; });
                }
                break;
                case 2: {
                    auto dst_codim_2d_span =
                        span{h_codim_2ds}.subspan(info.primitive_offset, info.primitive_count);

                    auto tri_view = sc->triangles().topo().view();
                    UIPC_ASSERT(tri_view.size() == dst_codim_2d_span.size(),
                                "triangle size mismatching");

                    std::transform(tri_view.begin(),
                                   tri_view.end(),
                                   dst_codim_2d_span.begin(),
                                   [&](const Vector3i& tri) -> Vector3i
                                   { return tri.array() + info.vertex_offset; });
                }
                break;
                case 3: {
                    auto dst_tet_span =
                        span{h_tets}.subspan(info.primitive_offset, info.primitive_count);

                    auto tet_view = sc->tetrahedra().topo().view();
                    UIPC_ASSERT(tet_view.size() == dst_tet_span.size(),
                                "tetrahedra size mismatching");

                    std::transform(tet_view.begin(),
                                   tet_view.end(),
                                   dst_tet_span.begin(),
                                   [&](const Vector4i& tet) -> Vector4i
                                   { return tet.array() + info.vertex_offset; });
                }
                break;
                default:
                    break;
            }

            {  // 2) fill backend_fem_vertex_offset in geometry
                auto vertex_offset =
                    sc->meta().find<IndexT>(builtin::backend_fem_vertex_offset);
                if(!vertex_offset)
                    vertex_offset =
                        sc->meta().create<IndexT>(builtin::backend_fem_vertex_offset, -1);
                auto vertex_offset_view = geometry::view(*vertex_offset);
                std::ranges::fill(vertex_offset_view, info.vertex_offset);
            }

            {  // 3) setup positions and velocities
                auto pos_view = sc->positions().view();
                auto dst_pos_span =
                    span{h_positions}.subspan(info.vertex_offset, info.vertex_count);
                UIPC_ASSERT(pos_view.size() == dst_pos_span.size(), "position size mismatching");
                std::copy(pos_view.begin(), pos_view.end(), dst_pos_span.begin());

                auto rest_pos_view = rest_sc->positions().view();
                auto dst_rest_pos_span =
                    span{h_rest_positions}.subspan(info.vertex_offset, info.vertex_count);
                UIPC_ASSERT(rest_pos_view.size() == dst_rest_pos_span.size(),
                            "rest position size mismatching");
                std::ranges::copy(rest_pos_view, dst_rest_pos_span.begin());

                auto vel = sc->vertices().find<Vector3>(builtin::velocity);
                if(vel)  // if user set the velocity
                {
                    auto vel_view = vel->view();
                    auto dst_vel_span =
                        span{h_velocities}.subspan(info.vertex_offset, info.vertex_count);
                    UIPC_ASSERT(vel_view.size() == dst_vel_span.size(),
                                "velocity size mismatching");
                    std::ranges::copy(vel_view, dst_vel_span.begin());
                }
                // else, keep the default value (0)
            }

            {  // 4) setup mass
                auto volume = rest_sc->vertices().find<Float>(builtin::volume);
                auto volume_view = volume->view();

                auto meta_mass_density = sc->meta().find<Float>(builtin::mass_density);
                auto vertex_mass_density = sc->vertices().find<Float>(builtin::mass_density);
                UIPC_ASSERT(meta_mass_density || vertex_mass_density,
                            "mass density is not found in the geometry");
                auto mass_density_view = vertex_mass_density ?
                                             vertex_mass_density->view() :
                                             meta_mass_density->view();

                auto dst_mass_span =
                    span{h_masses}.subspan(info.vertex_offset, info.vertex_count);
                UIPC_ASSERT(volume_view.size() == dst_mass_span.size(), "mass size mismatching");
                std::ranges::copy(volume_view, dst_mass_span.begin());

                for(auto&& [i, dst_vert_mass] : enumerate(dst_mass_span))
                {
                    auto density  = vertex_mass_density ? mass_density_view[i] :
                                                          mass_density_view[0];
                    dst_vert_mass = density * volume_view[i];
                }
            }

            {  // 5) setup thickness
                auto thickness = sc->vertices().find<Float>(builtin::thickness);
                auto dst_thickness_span =
                    span{h_thicknesses}.subspan(info.vertex_offset, info.vertex_count);

                if(thickness)
                {
                    auto thickness_view = thickness->view();
                    UIPC_ASSERT(thickness_view.size() == dst_thickness_span.size(),
                                "thickness size mismatching");
                    std::ranges::copy(thickness_view, dst_thickness_span.begin());
                }
            }

            {  // 6) setup vertex contact element id

                auto dst_eid_span = span{h_vertex_contact_element_ids}.subspan(
                    info.vertex_offset, info.vertex_count);

                auto vert_ceid = sc->vertices().find<IndexT>(builtin::contact_element_id);
                if(vert_ceid)
                {
                    auto ceid_view = vert_ceid->view();
                    UIPC_ASSERT(ceid_view.size() == dst_eid_span.size(),
                                "contact element id size mismatching");

                    std::ranges::copy(ceid_view, dst_eid_span.begin());
                }
                else
                {
                    auto ceid = sc->meta().find<IndexT>(builtin::contact_element_id);

                    if(ceid)
                    {
                        auto eid = ceid->view()[0];
                        std::ranges::fill(dst_eid_span, eid);
                    }
                }
            }

            {  // 7) setup vertex is_fixed

                auto is_fixed = sc->vertices().find<IndexT>(builtin::is_fixed);
                auto constraint_uid = sc->meta().find<U64>(builtin::constraint_uid);

                auto dst_is_fixed_span =
                    span{h_vertex_is_fixed}.subspan(info.vertex_offset, info.vertex_count);

                if(is_fixed)
                {
                    auto is_fixed_view = is_fixed->view();
                    UIPC_ASSERT(is_fixed_view.size() == dst_is_fixed_span.size(),
                                "is_fixed size mismatching");
                    std::ranges::copy(is_fixed_view, dst_is_fixed_span.begin());
                }
            }

            {  // 8) setup dimension
                auto dst_dim_span =
                    span{h_dimensions}.subspan(info.vertex_offset, info.vertex_count);
                std::ranges::fill(dst_dim_span, sc->dim());
            }

            {  // 9) setup vertex is_dynamic
                auto is_dynamic = sc->vertices().find<IndexT>(builtin::is_dynamic);
                auto dst_is_dynamic =
                    span{h_vertex_is_dynamic}.subspan(info.vertex_offset, info.vertex_count);

                if(is_dynamic)
                {
                    auto is_dynamic_view = is_dynamic->view();
                    UIPC_ASSERT(is_dynamic_view.size() == dst_is_dynamic.size(),
                                "is_kinematic size mismatching");
                    std::ranges::copy(is_dynamic_view, dst_is_dynamic.begin());
                }
            }

            {  // 10) setup vertex gravities

                auto gravity_attr = sc->vertices().find<Vector3>(builtin::gravity);
                auto dst_gravties =
                    span{h_gravities}.subspan(info.vertex_offset, info.vertex_count);

                if(gravity_attr)
                {
                    auto gravity_view = gravity_attr->view();
                    UIPC_ASSERT(gravity_view.size() == dst_gravties.size(),
                                "gravity size mismatching");
                    std::ranges::copy(gravity_view, dst_gravties.begin());
                }
            }

            {  // 11) setup vertex body id

                IndexT body_id = i;  // geo slot index is the body id
                auto   dst_body_id_span =
                    span{h_vertex_body_id}.subspan(info.vertex_offset, info.vertex_count);
                std::ranges::fill(dst_body_id_span, body_id);
            }
        }
    }

    // 2. Body Attributes
    {
        h_body_self_collision.resize(geo_infos.size(), 1);  // fill 1 for default turn on self-collision

        for(auto&& [i, info] : enumerate(geo_infos))
        {
            auto& geo_slot = geo_slots[info.geo_slot_index];
            auto& geo      = geo_slot->geometry();
            auto* sc       = geo.as<geometry::SimplicialComplex>();
            UIPC_ASSERT(sc,
                        "The geometry is not a simplicial complex (it's {}). Why can it happen?",
                        geo.type());

            {  // 1) setup body self-collision

                auto self_collision = sc->meta().find<IndexT>(builtin::self_collision);
                UIPC_ASSERT(self_collision, "self_collision is not found in finite element `meta`, why can it happen?");
                h_body_self_collision[i] = self_collision->view()[0];
            }
        }
    }
}

void FiniteElementMethod::Impl::_build_on_device()
{
    using namespace muda;

    // 1) Vertex States
    xs.resize(h_positions.size());
    xs.view().copy_from(h_positions.data());

    x_bars.resize(h_rest_positions.size());
    x_bars.view().copy_from(h_rest_positions.data());

    x_temps  = xs;
    x_tildes = xs;
    x_prevs  = xs;

    is_fixed.resize(h_vertex_is_fixed.size());
    is_fixed.view().copy_from(h_vertex_is_fixed.data());

    is_dynamic.resize(h_vertex_is_dynamic.size());
    is_dynamic.view().copy_from(h_vertex_is_dynamic.data());

    gravities.resize(h_gravities.size());
    gravities.view().copy_from(h_gravities.data());

    dxs.resize(xs.size(), Vector3::Zero());
    vs.resize(h_velocities.size());
    vs.view().copy_from(h_velocities.data());

    masses.resize(h_masses.size());
    masses.view().copy_from(h_masses.data());

    thicknesses.resize(h_thicknesses.size());
    thicknesses.view().copy_from(h_thicknesses.data());

    // 2) Elements
    codim_0ds.resize(h_codim_0ds.size());
    codim_0ds.view().copy_from(h_codim_0ds.data());

    codim_1ds.resize(h_codim_1ds.size());
    codim_1ds.view().copy_from(h_codim_1ds.data());
    rest_lengths.resize(codim_1ds.size());

    codim_2ds.resize(h_codim_2ds.size());
    codim_2ds.view().copy_from(h_codim_2ds.data());
    rest_areas.resize(codim_2ds.size());

    tets.resize(h_tets.size());
    tets.view().copy_from(h_tets.data());
    rest_volumes.resize(tets.size());

    // 3) Material Space Attribute
    // Rod
    ParallelFor()
        .kernel_name("Rod Basis")
        .apply(codim_1ds.size(),
               [codim_1ds = codim_1ds.viewer().name("codim_1ds"),
                x_bars    = x_bars.viewer().name("x_bars"),
                rest_lengths = rest_lengths.viewer().name("rest_lengths")] __device__(int i) mutable
               {
                   const Vector2i& edge = codim_1ds(i);
                   const Vector3&  x0   = x_bars(edge[0]);
                   const Vector3&  x1   = x_bars(edge[1]);

                   rest_lengths(i) = (x1 - x0).norm();
               });


    // Shell
    ParallelFor()
        .kernel_name("Shell Basis")
        .apply(codim_2ds.size(),
               [codim_2ds = codim_2ds.viewer().name("codim_2ds"),
                x_bars    = x_bars.viewer().name("x_bars"),
                rest_areas = rest_areas.viewer().name("rest_areas")] __device__(int i) mutable
               {
                   const Vector3i& tri = codim_2ds(i);
                   const Vector3&  x0  = x_bars(tri[0]);
                   const Vector3&  x1  = x_bars(tri[1]);
                   const Vector3&  x2  = x_bars(tri[2]);

                   Vector3 E01 = x1 - x0;
                   Vector3 E02 = x2 - x0;

                   rest_areas(i) = 0.5 * E01.cross(E02).norm();
               });

    // FEM3D Material Basis
    Dm3x3_invs.resize(tets.size());
    ParallelFor()
        .kernel_name("FEM3D Material Basis")
        .apply(tets.size(),
               [tets      = tets.viewer().name("tets"),
                x_bars    = x_bars.viewer().name("x_bars"),
                Dm9x9_inv = Dm3x3_invs.viewer().name("Dm3x3_inv"),
                rest_volumes = rest_volumes.viewer().name("rest_volumes")] __device__(int i) mutable
               {
                   const Vector4i& tet = tets(i);
                   const Vector3&  x0  = x_bars(tet[0]);
                   const Vector3&  x1  = x_bars(tet[1]);
                   const Vector3&  x2  = x_bars(tet[2]);
                   const Vector3&  x3  = x_bars(tet[3]);

                   Dm9x9_inv(i) = fem::Dm_inv(x0, x1, x2, x3);
                   Float V      = fem::Ds(x0, x1, x2, x3).determinant();
                   MUDA_ASSERT(V > 0.0,
                               "Negative volume tetrahedron (%d, %d, %d, %d)",
                               tet[0],
                               tet[1],
                               tet[2],
                               tet[3]);
                   rest_volumes(i) = V;
               });
}

void FiniteElementMethod::Impl::_init_base_constitution()
{
    for(auto&& [i, c] : enumerate(codim_0d_constitutions))
    {
        c->init();
    }

    for(auto&& [i, c] : enumerate(codim_1d_constitutions))
    {
        c->init();
    }

    for(auto&& [i, c] : enumerate(codim_2d_constitutions))
    {
        c->init();
    }

    for(auto&& [i, c] : enumerate(fem_3d_constitutions))
    {
        c->init();
    }
}

void FiniteElementMethod::Impl::_init_extra_constitutions()
{
    for(auto&& [i, c] : enumerate(extra_constitutions.view()))
    {
        c->init();
        auto uid = c->uid();
        extra_constitution_uid_to_index.insert({uid, i});
    }
}

void FiniteElementMethod::Impl::_init_energy_producers()
{
    auto constitution_view       = constitutions.view();
    auto extra_constitution_view = extra_constitutions.view();
    SizeT N = constitution_view.size() + constitution_view.size() + 1 /*Kinetic*/;
    energy_producers.reserve(N);
    energy_producers.push_back(kinetic.view());
    std::ranges::copy(constitution_view, std::back_inserter(energy_producers));
    std::ranges::copy(extra_constitution_view, std::back_inserter(energy_producers));

    // +1 for total count
    vector<SizeT> energy_counts(N + 1, 0);
    vector<SizeT> energy_offsets(N + 1, 0);
    vector<SizeT> gradient_counts(N + 1, 0);
    vector<SizeT> gradient_offsets(N + 1, 0);
    vector<SizeT> hessian_counts(N + 1, 0);
    vector<SizeT> hessian_offsets(N + 1, 0);

    for(auto&& [i, c] : enumerate(energy_producers))
    {
        c->collect_extent_info();
    }

    for(auto&& [i, c] : enumerate(energy_producers))
    {
        energy_counts[i]   = c->m_impl.energy_count;
        gradient_counts[i] = c->m_impl.gradient_count;
        hessian_counts[i]  = c->m_impl.hessian_count;
    }

    std::exclusive_scan(
        energy_counts.begin(), energy_counts.end(), energy_offsets.begin(), 0);
    std::exclusive_scan(
        gradient_counts.begin(), gradient_counts.end(), gradient_offsets.begin(), 0);
    std::exclusive_scan(
        hessian_counts.begin(), hessian_counts.end(), hessian_offsets.begin(), 0);

    for(auto&& [i, c] : enumerate(energy_producers))
    {
        c->m_impl.energy_offset   = energy_offsets[i];
        c->m_impl.gradient_offset = gradient_offsets[i];
        c->m_impl.hessian_offset  = hessian_offsets[i];
    }

    auto vertex_count   = xs.size();
    auto energy_count   = energy_offsets.back();
    auto gradient_count = gradient_offsets.back();

    energy_producer_energies.resize(energy_count);
    energy_producer_gradients.resize(vertex_count, gradient_count);
    energy_producer_total_hessian_count = hessian_offsets.back();
}

void FiniteElementMethod::Impl::_download_geometry_to_host()
{
    xs.view().copy_to(h_positions.data());
}

void FiniteElementMethod::Impl::write_scene(WorldVisitor& world)
{
    _download_geometry_to_host();

    auto geo_slots = world.scene().geometries();

    auto position_span = span{h_positions};

    for(auto&& [i, info] : enumerate(geo_infos))
    {
        auto& geo_slot = geo_slots[info.geo_slot_index];
        auto& geo      = geo_slot->geometry();
        auto* sc       = geo.as<geometry::SimplicialComplex>();
        UIPC_ASSERT(sc,
                    "The geometry is not a simplicial complex (it's {}). Why can it happen?",
                    geo.type());

        // 1) write positions back
        auto pos_view = geometry::view(sc->positions());
        auto src_pos_span = position_span.subspan(info.vertex_offset, info.vertex_count);
        UIPC_ASSERT(pos_view.size() == src_pos_span.size(), "position size mismatching");
        std::copy(src_pos_span.begin(), src_pos_span.end(), pos_view.begin());

        // 2) write primitives back
        // TODO:
        // Now there is no topology modification, so no need to write back
        // In the future, we may need to write back the topology if the topology is modified
    }
}
}  // namespace uipc::backend::cuda


// Info:
namespace uipc::backend::cuda
{
Float FiniteElementMethod::ComputeEnergyInfo::dt() const noexcept
{
    return m_dt;
}

FiniteElementMethod::ComputeGradientHessianInfo::ComputeGradientHessianInfo(Float dt) noexcept
    : m_dt(dt)
{
}
}  // namespace uipc::backend::cuda


// Dump & Recover:
namespace uipc::backend::cuda
{
bool FiniteElementMethod::Impl::dump(DumpInfo& info)
{
    auto path  = info.dump_path(__FILE__);
    auto frame = info.frame();

    return dump_xs.dump(fmt::format("{}q.{}", path, frame), xs)       //
           && dump_vs.dump(fmt::format("{}q_v.{}", path, frame), vs)  //
           && dump_x_prevs.dump(fmt::format("{}q_prev.{}", path, frame), x_prevs);  //
}

bool FiniteElementMethod::Impl::try_recover(RecoverInfo& info)
{
    auto path  = info.dump_path(__FILE__);
    auto frame = info.frame();

    return dump_xs.load(fmt::format("{}q.{}", path, frame))                //
           && dump_vs.load(fmt::format("{}q_v.{}", path, frame))           //
           && dump_x_prevs.load(fmt::format("{}q_prev.{}", path, frame));  //
}

void FiniteElementMethod::Impl::apply_recover(RecoverInfo& info)
{
    dump_xs.apply_to(xs);
    dump_vs.apply_to(vs);
    dump_x_prevs.apply_to(x_prevs);
}

void FiniteElementMethod::Impl::clear_recover(RecoverInfo& info)
{
    dump_xs.clean_up();
    dump_vs.clean_up();
    dump_x_prevs.clean_up();
}

void FiniteElementMethod::Impl::set_dof_info(SizeT frame, IndexT dof_offset, IndexT dof_count)
{
    UIPC_ASSERT(frame > 0, "frame 0 is not used");
    if(frame_to_dof_count.size() <= frame)
    {
        frame_to_dof_count.resize(frame + 1, -1);
        frame_to_dof_offset.resize(frame + 1, -1);
    }
    frame_to_dof_count[frame]  = dof_count;
    frame_to_dof_offset[frame] = dof_offset;
}

IndexT FiniteElementMethod::Impl::dof_offset(SizeT frame) const noexcept
{
    return frame_to_dof_offset[frame];
}

IndexT FiniteElementMethod::Impl::dof_count(SizeT frame) const noexcept
{
    return frame_to_dof_count[frame];
}


auto FiniteElementMethod::FilteredInfo::geo_infos() const noexcept -> span<const GeoInfo>
{
    auto info = this->constitution_info();
    return span{m_impl->geo_infos}.subspan(info.geo_info_offset, info.geo_info_count);
}


auto FiniteElementMethod::FilteredInfo::constitution_info() const noexcept
    -> const ConstitutionInfo&
{
    switch(m_dim)
    {
        case 0:
            return m_impl->codim_0d_constitution_infos[m_index_in_dim];
        case 1:
            return m_impl->codim_1d_constitution_infos[m_index_in_dim];
        case 2:
            return m_impl->codim_2d_constitution_infos[m_index_in_dim];
        case 3:
            return m_impl->fem_3d_constitution_infos[m_index_in_dim];
        default:
            UIPC_ASSERT(false, "Invalid dimension");
            return m_impl->codim_0d_constitution_infos[m_index_in_dim];  // dummy
    }
}

size_t FiniteElementMethod::FilteredInfo::vertex_count() const noexcept
{
    return constitution_info().vertex_count;
}

size_t FiniteElementMethod::FilteredInfo::primitive_count() const noexcept
{
    return constitution_info().primitive_count;
}
}  // namespace uipc::backend::cuda