#include "hip/hip_runtime.h"
#include <collision_detection/filters/easy_vertex_half_plane_trajectory_filter.h>
#include <muda/cub/device/device_reduce.h>
#include <kernel_cout.h>
#include <utils/codim_thickness.h>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(EasyVertexHalfPlaneTrajectoryFilter);

constexpr bool PrintDebugInfo = false;

void EasyVertexHalfPlaneTrajectoryFilter::do_detect(DetectInfo& info)
{
    // do nothing
}

void EasyVertexHalfPlaneTrajectoryFilter::do_filter_active(FilterActiveInfo& info)
{
    m_impl.filter_active(info);
}

void EasyVertexHalfPlaneTrajectoryFilter::do_filter_toi(FilterTOIInfo& info)
{
    m_impl.filter_toi(info);
}

void EasyVertexHalfPlaneTrajectoryFilter::Impl::filter_active(FilterActiveInfo& info)
{
    using namespace muda;

    auto query = [&]
    {
        num_collisions = 0;

        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(info.surf_vertices().size(),
                   [num = num_collisions.viewer().name("num_collisions"),
                    plane_vertex_offset = info.plane_vertex_global_offset(),
                    surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                    positions = info.positions().viewer().name("positions"),
                    thicknesses = info.thicknesses().viewer().name("thicknesses"),
                    contact_element_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                    contact_mask_tabular = info.contact_mask_tabular().viewer().name("contact_mask_tabular"),
                    half_plane_positions = info.plane_positions().viewer().name("plane_positions"),
                    half_plane_normals = info.plane_normals().viewer().name("plane_normals"),
                    d_hat     = info.d_hat(),
                    PHs       = PHs.viewer().name("PHs"),
                    max_count = PHs.size()] __device__(int i) mutable
                   {
                       for(int j = 0; j < half_plane_positions.total_size(); ++j)
                       {
                           IndexT vI = surf_vertices(i);
                           IndexT vJ = plane_vertex_offset + j;

                           IndexT L = contact_element_ids(vI);
                           IndexT R = contact_element_ids(vJ);

                           if(contact_mask_tabular(L, R) == 0)
                               continue;

                           Vector3 pos = positions(vI);

                           Vector3 plane_pos    = half_plane_positions(j);
                           Vector3 plane_normal = half_plane_normals(j);

                           Vector3 diff = pos - plane_pos;

                           Float dst = diff.dot(plane_normal);

                           Float thickness = thicknesses(vI);

                           Float D = dst * dst;

                           auto range = D_range(thickness, d_hat);

                           if(is_active_D(range, D))
                           {
                               auto last = atomic_add(num.data(), 1);

                               if(last < max_count)
                               {
                                   PHs(last) = Vector2i{vI, j};
                               }
                           }
                       }
                   });
    };

    query();
    h_num_collisions = num_collisions;

    if(h_num_collisions > PHs.size())
    {
        PHs.resize(h_num_collisions * reserve_ratio);
        query();
    }

    info.PHs(PHs.view(0, h_num_collisions));

    if constexpr(PrintDebugInfo)
    {
        std::vector<Vector2i> phs(h_num_collisions);
        PHs.view(0, h_num_collisions).copy_to(phs.data());
        for(auto& ph : phs)
        {
            std::cout << "vI: " << ph[0] << ", pI: " << ph[1] << std::endl;
        }
    }
}

void EasyVertexHalfPlaneTrajectoryFilter::Impl::filter_toi(FilterTOIInfo& info)
{
    using namespace muda;

    info.toi().fill(1.1f);
    tois.resize(info.surf_vertices().size());

    // TODO: just hard code the slackness for now
    constexpr Float eta = 0.1;

    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(info.surf_vertices().size(),
               [surf_vertices = info.surf_vertices().viewer().name("surf_vertices"),
                plane_vertex_offset = info.plane_vertex_global_offset(),
                positions   = info.positions().viewer().name("positions"),
                thicknesses = info.thicknesses().viewer().name("thicknesses"),
                contact_element_ids = info.contact_element_ids().viewer().name("contact_element_ids"),
                contact_mask_tabular = info.contact_mask_tabular().viewer().name("contact_mask_tabular"),
                displacements = info.displacements().viewer().name("displacements"),
                half_plane_positions = info.plane_positions().viewer().name("plane_positions"),
                half_plane_normals = info.plane_normals().viewer().name("plane_normals"),
                tois  = tois.viewer().name("tois"),
                alpha = info.alpha(),
                d_hat = info.d_hat(),
                eta] __device__(int i) mutable
               {
                   Float min_toi = 1.1f;  // large enough

                   for(int j = 0; j < half_plane_positions.total_size(); ++j)
                   {
                       IndexT vI = surf_vertices(i);
                       IndexT vJ = plane_vertex_offset + j;

                       IndexT L = contact_element_ids(vI);
                       IndexT R = contact_element_ids(vJ);

                       if(contact_mask_tabular(L, R) == 0)
                           continue;

                       Vector3 x   = positions(vI);
                       Vector3 dx  = displacements(vI) * alpha;
                       Vector3 x_t = x + dx;


                       Vector3 P = half_plane_positions(j);
                       Vector3 N = half_plane_normals(j);

                       Float thickness = thicknesses(vI);

                       Float t = -N.dot(dx);
                       if(t <= 0)  // moving away from the plane, no collision
                           continue;

                       // t > 0, moving towards the plane


                       Vector3 diff = P - x;
                       Float   t0   = -N.dot(diff) - thickness;

                       Float this_toi = t0 / t * (1 - eta);

                       min_toi = std::min(min_toi, this_toi);

                       //if constexpr(PrintDebugInfo)
                       //{
                       //    if(this_toi < 1.0)
                       //    {
                       //        cout << "vI: " << vI << ", pI: " << j
                       //             << ", toi: " << this_toi << " d0: " << -t0 << "\n";
                       //    }
                       //}
                   }

                   tois(i) = min_toi;
               });

    DeviceReduce().Min(tois.data(), info.toi().data(), info.surf_vertices().size());
}
}  // namespace uipc::backend::cuda
