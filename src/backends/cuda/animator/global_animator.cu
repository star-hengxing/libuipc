#include "hip/hip_runtime.h"
#include <animator/global_animator.h>
#include <animator/animator.h>
#include <uipc/builtin/constitution_type.h>
#include <sim_engine.h>

namespace uipc::backend
{
template <>
class backend::SimSystemCreator<cuda::GlobalAnimator>
{
  public:
    static U<cuda::GlobalAnimator> create(SimEngine& engine)
    {
        auto  scene = dynamic_cast<cuda::SimEngine&>(engine).world().scene();
        auto& types = scene.constitution_tabular().types();
        if(types.find(std::string{builtin::Constraint}) == types.end())
        {
            return nullptr;
        }
        return uipc::make_unique<cuda::GlobalAnimator>(engine);
    }
};
}  // namespace uipc::backend

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalAnimator);

void GlobalAnimator::do_build() {}

Float GlobalAnimator::substep_ratio() noexcept
{
    return m_substep_ratio;
}

void GlobalAnimator::init()
{
    // init frontend animator
    world().animator().init();

    // init backend animator
    for(auto&& animator : m_animators.view())
    {
        animator->init();
    }
}

void GlobalAnimator::step()
{
    // update frontend animator
    world().animator().update();

    // after frontend update, reset substep ratio
    // prepare for the next newton iteration
    m_substep_ratio = 0.0;

    // update backend animator
    for(auto&& animator : m_animators.view())
    {
        animator->step();
    }
}

void GlobalAnimator::compute_substep_ratio(SizeT newton_iter)
{
    Float substep = static_cast<Float>(world().animator().substep());
    Float t       = (static_cast<Float>(newton_iter) + Float{1.0}) / substep;
    UIPC_ASSERT(substep > 0, "substep must be greater than 0");
    m_substep_ratio = std::min(t, Float{1.0});  // clamp t to [0, 1]
}

void GlobalAnimator::register_animator(Animator* animator)
{
    m_animators.register_subsystem(*animator);
}
}  // namespace uipc::backend::cuda
