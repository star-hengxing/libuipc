#include "hip/hip_runtime.h"
#include <animator/global_animator.h>
#include <collision_detection/global_trajectory_filter.h>
#include <contact_system/global_contact_manager.h>
#include <diff_sim/global_diff_sim_manager.h>
#include <dof_predictor.h>
#include <fstream>
#include <global_geometry/global_simplicial_surface_manager.h>
#include <global_geometry/global_vertex_manager.h>
#include <gradient_hessian_computer.h>
#include <line_search/line_searcher.h>
#include <linear_system/global_linear_system.h>
#include <sim_engine.h>
#include <uipc/common/log.h>
#include <affine_body/affine_body_dynamics.h>
#include <finite_element/finite_element_method.h>
#include <global_geometry/global_body_manager.h>

namespace uipc::backend::cuda
{
void SimEngine::build()
{
    // 1) build all systems
    build_systems();

    // 2) find those engine-aware topo systems
    m_global_vertex_manager     = &require<GlobalVertexManager>();
    m_global_body_manager       = &require<GlobalBodyManager>();
    m_dof_predictor             = &require<DofPredictor>();
    m_line_searcher             = &require<LineSearcher>();
    m_gradient_hessian_computer = &require<GradientHessianComputer>();
    m_global_linear_system      = &require<GlobalLinearSystem>();

    m_global_simplicial_surface_manager = find<GlobalSimpicialSurfaceManager>();
    m_global_contact_manager            = find<GlobalContactManager>();
    m_global_trajectory_filter          = find<GlobalTrajectoryFilter>();
    m_global_animator                   = find<GlobalAnimator>();
    m_global_diff_sim_manager           = find<GlobalDiffSimManager>();

    m_affine_body_dynamics  = find<AffineBodyDynamics>();
    m_finite_element_method = find<FiniteElementMethod>();

    // 3) dump system info
    dump_system_info();
}

void SimEngine::init_scene()
{
    auto& info            = world().scene().info();
    m_newton_velocity_tol = info["newton"]["velocity_tol"];
    m_newton_max_iter     = info["newton"]["max_iter"];
    m_ccd_tol             = info["newton"]["ccd_tol"];
    m_friction_enabled    = info["contact"]["friction"]["enable"];
    m_strict_mode         = info["extras"]["strict_mode"]["enable"];
    Vector3 gravity       = info["gravity"];
    Float   dt            = info["dt"];

    m_abs_tol = m_newton_velocity_tol * dt;

    // 1. Before Common Scene Initialization
    if(m_affine_body_dynamics)
        m_affine_body_dynamics->init();
    if(m_finite_element_method)
        m_finite_element_method->init();
    m_global_body_manager->init();

    // 2. Common Scene Initialization Phase
    event_init_scene();

    // 3. After Common Scene Initialization
    // 3.1 Forwards
    m_global_vertex_manager->init();
    m_global_simplicial_surface_manager->init();

    if(m_global_contact_manager)
        m_global_contact_manager->init();
    if(m_global_animator)
        m_global_animator->init();
    m_global_linear_system->init();

    // 3.2 Backwards (if needed)
    if(m_global_diff_sim_manager)
        m_global_diff_sim_manager->init();
    //if(m_global_diff_contact_manager)
    //    m_global_diff_contact_manager->init();
    //if(m_abd_diff_sim_manager)
    //    m_abd_diff_sim_manager->init();
}

void SimEngine::do_init(InitInfo& info)
{
    try
    {
        // 1. Build all the systems and their dependencies
        m_state = SimEngineState::BuildSystems;
        build();

        // 2. Trigger the init_scene event, systems register their actions will be called here
        m_state = SimEngineState::InitScene;
        init_scene();

        // 3. Any creation and deletion of objects after this point will be pending
        world().scene().begin_pending();
    }
    catch(const SimEngineException& e)
    {
        spdlog::error("SimEngine init error: {}", e.what());
        status().push_back(core::EngineStatus::error(e.what()));
    }
}
}  // namespace uipc::backend::cuda