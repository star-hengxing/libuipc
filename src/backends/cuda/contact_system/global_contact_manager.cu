#include "hip/hip_runtime.h"
#include <contact_system/global_contact_manager.h>
#include <collision_detection/global_trajectory_filter.h>
#include <sim_engine.h>
#include <contact_system/contact_reporter.h>
#include <contact_system/contact_receiver.h>
#include <uipc/common/enumerate.h>
#include <kernel_cout.h>
#include <uipc/common/unit.h>
#include <uipc/common/zip.h>

namespace uipc::backend
{
template <>
class SimSystemCreator<cuda::GlobalContactManager>
{
  public:
    static U<cuda::GlobalContactManager> create(cuda::SimEngine& engine)
    {
        if(engine.world().scene().info()["contact"]["enable"])
            return make_unique<cuda::GlobalContactManager>(engine);
        return nullptr;
    }
};
}  // namespace uipc::backend

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalContactManager);

void GlobalContactManager::do_build()
{
    const auto& info = world().scene().info();

    m_impl.global_vertex_manager    = require<GlobalVertexManager>();
    m_impl.global_trajectory_filter = require<GlobalTrajectoryFilter>();


    m_impl.d_hat        = info["contact"]["d_hat"].get<Float>();
    m_impl.dt           = info["dt"].get<Float>();
    m_impl.eps_velocity = info["contact"]["eps_velocity"].get<Float>();
    m_impl.cfl_enabled  = info["cfl"]["enable"].get<bool>();
    m_impl.kappa = world().scene().contact_tabular().default_model().resistance();
}

muda::CBuffer2DView<IndexT> GlobalContactManager::contact_mask_tabular() const noexcept
{
    return m_impl.contact_mask_tabular;
}

void GlobalContactManager::Impl::init(WorldVisitor& world)
{
    // 1) init tabular
    auto contact_models = world.scene().contact_tabular().contact_models();

    auto attr_topo          = contact_models.find<Vector2i>("topo");
    auto attr_resistance    = contact_models.find<Float>("resistance");
    auto attr_friction_rate = contact_models.find<Float>("friction_rate");
    auto attr_enabled       = contact_models.find<IndexT>("is_enabled");

    UIPC_ASSERT(attr_topo != nullptr, "topo is not found in contact tabular");
    UIPC_ASSERT(attr_resistance != nullptr, "resistance is not found in contact tabular");
    UIPC_ASSERT(attr_friction_rate != nullptr, "friction_rate is not found in contact tabular");
    UIPC_ASSERT(attr_enabled != nullptr, "is_enabled is not found in contact tabular");

    auto topo_view          = attr_topo->view();
    auto resistance_view    = attr_resistance->view();
    auto friction_rate_view = attr_friction_rate->view();
    auto enabled_view       = attr_enabled->view();

    auto N = world.scene().contact_tabular().element_count();

    h_contact_tabular.resize(
        N * N, ContactCoeff{.kappa = resistance_view[0], .mu = friction_rate_view[0]});

    h_contact_mask_tabular.resize(N * N, 1);

    for(auto&& [ids, kappa, mu, is_enabled] :
        zip(topo_view, resistance_view, friction_rate_view, enabled_view))
    {

        ContactCoeff coeff{.kappa = kappa, .mu = mu};

        auto upper                    = ids.x() * N + ids.y();
        h_contact_tabular[upper]      = coeff;
        h_contact_mask_tabular[upper] = is_enabled;

        auto lower                    = ids.y() * N + ids.x();
        h_contact_tabular[lower]      = coeff;
        h_contact_mask_tabular[lower] = is_enabled;
    }

    // print table:
    // for(auto i = 0; i < N; ++i)
    // {
    //     for(auto j = 0; j < N; ++j)
    //     {
    //         auto idx   = i * N + j;
    //         auto coeff = h_contact_tabular[idx];
    //         auto mask  = h_contact_mask_tabular[idx];
    //         std::cout << "(" << i << ", " << j << ") : " << coeff.kappa << ", "
    //                   << coeff.mu << ", " << mask << "\n";
    //     }
    // }

    contact_tabular.resize(muda::Extent2D{N, N});
    contact_tabular.view().copy_from(h_contact_tabular.data());

    contact_mask_tabular.resize(muda::Extent2D{N, N});
    contact_mask_tabular.view().copy_from(h_contact_mask_tabular.data());

    // muda::Launch().apply(
    //     [contact_mask_tabular = contact_mask_tabular.viewer().name("contact_mask_tabular"),
    //      N] __device__()
    //     {
    //         for(auto i = 0; i < N; ++i)
    //         {
    //             for(auto j = 0; j < N; ++j)
    //             {
    //                 auto idx   = i * N + j;
    //                 auto coeff = contact_mask_tabular(i, j);
    //                 printf("%d %d = %d\n", i, j, coeff);
    //             }
    //         }
    //     });

    // 2) vertex contact info
    vert_is_active_contact.resize(global_vertex_manager->positions().size(), 0);
    vert_disp_norms.resize(global_vertex_manager->positions().size(), 0.0);

    // 3) reporters
    auto contact_reporter_view = contact_reporters.view();
    for(auto&& [i, R] : enumerate(contact_reporter_view))
        R->m_index = i;

    reporter_gradient_offsets.resize(contact_reporter_view.size());
    reporter_gradient_counts.resize(contact_reporter_view.size());

    reporter_hessian_offsets.resize(contact_reporter_view.size());
    reporter_hessian_counts.resize(contact_reporter_view.size());

    // 4) receivers
    auto contact_receiver_view = contact_receivers.view();
    for(auto&& [i, R] : enumerate(contact_receiver_view))
        R->m_index = i;

    classified_contact_gradients.resize(contact_receiver_view.size());
    classified_contact_hessians.resize(contact_receiver_view.size());
}

void GlobalContactManager::Impl::compute_d_hat()
{
    // TODO: Now do nothing
}

void GlobalContactManager::Impl::compute_adaptive_kappa()
{
    // TODO: Now do nothing
}

Float GlobalContactManager::Impl::compute_cfl_condition()
{
    if(!cfl_enabled)  // if cfl is disabled, just return 1.0
        return 1.0;

    vert_is_active_contact.fill(0);  // clear the active flag

    global_trajectory_filter->label_active_vertices();

    auto displacements = global_vertex_manager->displacements();

    using namespace muda;
    ParallelFor()
        .file_line(__FILE__, __LINE__)
        .apply(displacements.size(),
               [disps             = displacements.cviewer().name("disp"),
                disp_norms        = vert_disp_norms.viewer().name("disp_norm"),
                is_contact_active = vert_is_active_contact.viewer().name(
                    "vert_is_contact_active")] __device__(int i) mutable
               {
                   // if the contact is not active, then the displacement is ignored
                   disp_norms(i) = is_contact_active(i) ? disps(i).norm() : 0.0;
               });

    DeviceReduce().Max(
        vert_disp_norms.data(), max_disp_norm.data(), vert_disp_norms.size());

    Float h_max_disp_norm = max_disp_norm;
    return h_max_disp_norm == 0.0 ? 1.0 : std::min(0.5 * d_hat / h_max_disp_norm, 1.0);
}

void GlobalContactManager::Impl::compute_contact()
{
    _assemble();
    _convert_matrix();
    _distribute();
}

void GlobalContactManager::Impl::_assemble()
{
    auto vertex_count = global_vertex_manager->positions().size();

    for(auto&& [i, reporter] : enumerate(contact_reporters.view()))
    {
        ContactExtentInfo info;
        reporter->report_extent(info);
        reporter_gradient_counts[i] = info.m_gradient_count;
        reporter_hessian_counts[i]  = info.m_hessian_count;
        spdlog::info("<{}> contact Grad3 count: {}, contact Hess3x3 count: {}",
                     reporter->name(),
                     info.m_gradient_count,
                     info.m_hessian_count);
    }

    // scan
    std::exclusive_scan(reporter_gradient_counts.begin(),
                        reporter_gradient_counts.end(),
                        reporter_gradient_offsets.begin(),
                        0);
    std::exclusive_scan(reporter_hessian_counts.begin(),
                        reporter_hessian_counts.end(),
                        reporter_hessian_offsets.begin(),
                        0);

    auto total_gradient_count =
        reporter_gradient_offsets.back() + reporter_gradient_counts.back();
    auto total_hessian_count =
        reporter_hessian_offsets.back() + reporter_hessian_counts.back();

    // allocate
    loose_resize_entries(collected_contact_gradient, total_gradient_count);
    loose_resize_entries(sorted_contact_gradient, total_gradient_count);
    loose_resize_entries(collected_contact_hessian, total_hessian_count);
    loose_resize_entries(sorted_contact_hessian, total_hessian_count);
    collected_contact_gradient.reshape(vertex_count);
    collected_contact_hessian.reshape(vertex_count, vertex_count);

    // collect
    for(auto&& [i, reporter] : enumerate(contact_reporters.view()))
    {
        auto g_offset = reporter_gradient_offsets[i];
        auto g_count  = reporter_gradient_counts[i];
        auto h_offset = reporter_hessian_offsets[i];
        auto h_count  = reporter_hessian_counts[i];

        ContactInfo info;

        info.m_gradient = collected_contact_gradient.view().subview(g_offset, g_count);
        info.m_hessian = collected_contact_hessian.view().subview(h_offset, h_count);

        reporter->assemble(info);
    }
}

void GlobalContactManager::Impl::_convert_matrix()
{
    matrix_converter.convert(collected_contact_hessian, sorted_contact_hessian);
    matrix_converter.convert(collected_contact_gradient, sorted_contact_gradient);
}

void GlobalContactManager::Impl::_distribute()
{
    using namespace muda;

    auto vertex_count = global_vertex_manager->positions().size();

    for(auto&& [i, receiver] : enumerate(contact_receivers.view()))
    {
        ClassifyInfo info;
        receiver->report(info);

        auto& classified_gradients = classified_contact_gradients[i];
        classified_gradients.reshape(vertex_count);
        auto& classified_hessians = classified_contact_hessians[i];
        classified_hessians.reshape(vertex_count, vertex_count);

        // 1) report gradient
        if(info.is_diag())
        {
            const auto N = sorted_contact_gradient.doublet_count();

            // clear the range in device
            gradient_range = Vector2i{0, 0};

            // partition
            ParallelFor()
                .kernel_name(__FUNCTION__)
                .apply(N,
                       [gradient_range = gradient_range.viewer().name("gradient_range"),
                        contact_gradient =
                            std::as_const(sorted_contact_gradient).viewer().name("contact_gradient"),
                        range = info.m_gradient_i_range] __device__(int I) mutable
                       {
                           auto in_range = [](int i, const Vector2i& range)
                           { return i >= range.x() && i < range.y(); };

                           auto&& [i, G]      = contact_gradient(I);
                           bool this_in_range = in_range(i, range);

                           //cout << "I: " << I << ", i: " << i << ", G: " << G
                           //     << ", in_range: " << this_in_range << "\n";

                           if(!this_in_range)
                           {
                               return;
                           }

                           bool prev_in_range = false;
                           if(I > 0)
                           {
                               auto&& [prev_i, prev_G] = contact_gradient(I - 1);
                               prev_in_range = in_range(prev_i, range);
                           }
                           bool next_in_range = false;
                           if(I < contact_gradient.total_doublet_count() - 1)
                           {
                               auto&& [next_i, next_G] = contact_gradient(I + 1);
                               next_in_range = in_range(next_i, range);
                           }

                           // if the prev is not in range, then this is the start of the partition
                           if(!prev_in_range)
                           {
                               gradient_range->x() = I;
                           }
                           // if the next is not in range, then this is the end of the partition
                           if(!next_in_range)
                           {
                               gradient_range->y() = I + 1;
                           }
                       });

            Vector2i h_range = gradient_range;  // copy back

            auto count = h_range.y() - h_range.x();

            loose_resize_entries(classified_gradients, count);

            // fill
            if(count > 0)
            {
                ParallelFor()
                    .kernel_name(__FUNCTION__)
                    .apply(count,
                           [contact_gradient =
                                std::as_const(sorted_contact_gradient).viewer().name("contact_gradient"),
                            classified_gradient = classified_gradients.viewer().name("classified_gradient"),
                            range = h_range] __device__(int I) mutable
                           {
                               auto&& [i, G] = contact_gradient(range.x() + I);
                               classified_gradient(I).write(i, G);
                           });
            }
        }

        // 2) report hessian
        if(!info.is_empty())
        {
            const auto N = sorted_contact_hessian.triplet_count();

            // +1 for calculate the total count
            loose_resize(selected_hessian, N + 1);
            loose_resize(selected_hessian_offsets, N + 1);

            // select
            ParallelFor()
                .kernel_name(__FUNCTION__)
                .apply(
                    N,
                    [selected_hessian = selected_hessian.view(0, N).viewer().name("selected_hessian"),
                     last =
                         VarView<IndexT>{selected_hessian.data() + N}.viewer().name("last"),
                     contact_hessian = sorted_contact_hessian.cviewer().name("contact_hessian"),
                     i_range = info.m_hessian_i_range,
                     j_range = info.m_hessian_j_range] __device__(int I) mutable
                    {
                        auto&& [i, j, H] = contact_hessian(I);

                        auto in_range = [](int i, const Vector2i& range)
                        { return i >= range.x() && i < range.y(); };

                        selected_hessian(I) =
                            in_range(i, i_range) && in_range(j, j_range) ? 1 : 0;

                        // fill the last one as 0, so that we can calculate the total count
                        // during the exclusive scan
                        if(I == 0)
                            last = 0;
                    });

            // scan
            DeviceScan().ExclusiveSum(selected_hessian.data(),
                                      selected_hessian_offsets.data(),
                                      selected_hessian.size());

            IndexT h_total_count = 0;
            VarView<IndexT>{selected_hessian_offsets.data() + N}.copy_to(&h_total_count);

            loose_resize_entries(classified_hessians, h_total_count);

            // fill
            if(h_total_count > 0)
            {
                ParallelFor()
                    .kernel_name(__FUNCTION__)
                    .apply(N,
                           [selected_hessian = selected_hessian.cviewer().name("selected_hessian"),
                            selected_hessian_offsets =
                                selected_hessian_offsets.cviewer().name("selected_hessian_offsets"),
                            contact_hessian = sorted_contact_hessian.cviewer().name("contact_hessian"),
                            classified_hessian = classified_hessians.viewer().name("classified_hessian"),
                            i_range = info.m_hessian_i_range,
                            j_range = info.m_hessian_j_range] __device__(int I) mutable
                           {
                               if(selected_hessian(I))
                               {
                                   auto&& [i, j, H] = contact_hessian(I);
                                   auto offset = selected_hessian_offsets(I);

                                   classified_hessian(offset).write(i, j, H);
                               }
                           });
            }

            ClassifiedContactInfo classified_info;

            classified_info.m_gradient = classified_gradients.view();
            classified_info.m_hessian  = classified_hessians.view();

            receiver->receive(classified_info);
        }
    }
}

void GlobalContactManager::Impl::loose_resize_entries(muda::DeviceTripletMatrix<Float, 3>& m,
                                                      SizeT size)
{
    if(size > m.triplet_capacity())
    {
        m.reserve_triplets(size * reserve_ratio);
    }
    m.resize_triplets(size);
}

void GlobalContactManager::Impl::loose_resize_entries(muda::DeviceDoubletVector<Float, 3>& v,
                                                      SizeT size)
{
    if(size > v.doublet_capacity())
    {
        v.reserve_doublets(size * reserve_ratio);
    }
    v.resize_doublets(size);
}


void GlobalContactManager::ClassifyInfo::range(const Vector2i& LRange, const Vector2i& RRange)
{
    m_type             = Type::Range;
    m_hessian_i_range  = LRange;
    m_hessian_j_range  = RRange;
    m_gradient_i_range = Vector2i::Zero();
}

void GlobalContactManager::ClassifyInfo::range(const Vector2i& Range)
{
    m_type             = Type::Range;
    m_gradient_i_range = Range;
    m_hessian_i_range  = Range;
    m_hessian_j_range  = Range;
}

bool GlobalContactManager::ClassifyInfo::is_empty() const
{
    return m_hessian_i_range[0] == m_hessian_i_range[1]
           || m_hessian_j_range[0] == m_hessian_j_range[1];
}

bool GlobalContactManager::ClassifyInfo::is_diag() const
{
    return m_gradient_i_range[0] != m_gradient_i_range[1];
}

void GlobalContactManager::ClassifyInfo::sanity_check()
{
    if(is_diag())
    {
        UIPC_ASSERT(m_gradient_i_range.x() <= m_gradient_i_range.y(),
                    "Diagonal Contact Gradient Range is invalid, [{}, {})",
                    m_gradient_i_range.x(),
                    m_gradient_i_range.y());

        UIPC_ASSERT(m_hessian_i_range == m_hessian_j_range,
                    "Diagonal Contact Hessian must have the same i_range and j_range");
    }
    else
    {
        UIPC_ASSERT(m_gradient_i_range.x() == m_gradient_i_range.y(),
                    "Off-Diagonal Contact must not have Gradient Part");
    }

    UIPC_ASSERT(m_hessian_i_range.x() <= m_hessian_i_range.y(),
                "Contact Hessian Range-i is invalid");
    UIPC_ASSERT(m_hessian_j_range.x() <= m_hessian_j_range.y(),
                "Contact Hessian Range-j is invalid");
}
}  // namespace uipc::backend::cuda


namespace uipc::backend::cuda
{
void GlobalContactManager::compute_d_hat()
{
    m_impl.compute_d_hat();
}

void GlobalContactManager::compute_contact()
{
    m_impl.compute_contact();
}

void GlobalContactManager::compute_adaptive_kappa()
{
    m_impl.compute_adaptive_kappa();
}

Float GlobalContactManager::compute_cfl_condition()
{
    return m_impl.compute_cfl_condition();
}

void GlobalContactManager::init()
{
    m_impl.init(world());
}

Float GlobalContactManager::d_hat() const
{
    return m_impl.d_hat;
}
Float GlobalContactManager::eps_velocity() const
{
    return m_impl.eps_velocity;
}
bool GlobalContactManager::cfl_enabled() const
{
    return m_impl.cfl_enabled;
}
void GlobalContactManager::add_reporter(ContactReporter* reporter)
{
    check_state(SimEngineState::BuildSystems, "add_reporter()");
    UIPC_ASSERT(reporter != nullptr, "reporter is nullptr");
    m_impl.contact_reporters.register_subsystem(*reporter);
}
void GlobalContactManager::add_receiver(ContactReceiver* receiver)
{
    check_state(SimEngineState::BuildSystems, "add_receiver()");
    UIPC_ASSERT(receiver != nullptr, "receiver is nullptr");
    m_impl.contact_receivers.register_subsystem(*receiver);
}
muda::CBuffer2DView<ContactCoeff> GlobalContactManager::contact_tabular() const noexcept
{
    return m_impl.contact_tabular;
}
}  // namespace uipc::backend::cuda