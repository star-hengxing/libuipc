#include "hip/hip_runtime.h"
#include <linear_system/spmv.h>
#include <muda/launch/launch.h>
#include <cub/warp/warp_reduce.cuh>
#include <cub/warp/warp_scan.cuh>
#include <cub/util_math.cuh>

namespace uipc::backend::cuda
{
void Spmv::sym_spmv(Float                           a,
                    muda::CBCOOMatrixView<Float, 3> A,
                    muda::CDenseVectorView<Float>   x,
                    Float                           b,
                    muda::DenseVectorView<Float>    y)
{

    constexpr int N = 3;
    using T         = Float;

    if(b != 0)
    {
        muda::ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(y.size(),
                   [b = b, y = y.viewer().name("y")] __device__(int i) mutable
                   { y(i) = b * y(i); });
    }
    else
    {
        muda::BufferLaunch().fill<Float>(y.buffer_view(), 0);
    }

    muda::ParallelFor()
        .kernel_name(__FUNCTION__)
        .apply(A.triplet_count(),
               [a = a,
                A = A.viewer().name("A"),
                x = x.viewer().name("x"),
                b = b,
                y = y.viewer().name("y")] __device__(int index) mutable
               {
                   auto&& [i, j, block] = A(index);

                   if(i == j)  // diagonal block
                   {
                       auto seg_x = x.segment<N>(j * N);

                       Eigen::Vector<T, N> vec_x  = seg_x.as_eigen();
                       auto                result = a * block * vec_x;

                       auto seg_y = y.segment<N>(i * N);
                       seg_y.atomic_add(result.eval());
                   }
                   else  // off-diagonal block
                   {
                       // ij-th block
                       {
                           auto seg_x = x.segment<N>(j * N);

                           Eigen::Vector<T, N> vec_x  = seg_x.as_eigen();
                           auto                result = a * block * vec_x;

                           auto seg_y = y.segment<N>(i * N);
                           seg_y.atomic_add(result.eval());
                       }

                       // ji-th block
                       {
                           auto seg_x = x.segment<N>(i * N);

                           Eigen::Vector<T, N> vec_x = seg_x.as_eigen();
                           auto result = a * block.transpose() * vec_x;

                           auto seg_y = y.segment<N>(j * N);
                           seg_y.atomic_add(result.eval());
                       }
                   }
               });
}

__host__ __device__ constexpr int b2i(bool b)
{
    return b ? 1 : 0;
}

struct Flags
{
    union
    {
        struct
        {
            unsigned char is_head;
            unsigned char is_cross_warp;
            unsigned char is_valid;
        };
        unsigned int flags;
    };

    __host__ __device__ void b2i()
    {
        is_head       = is_head ? 1 : 0;
        is_cross_warp = is_cross_warp ? 1 : 0;
        is_valid      = is_valid ? 1 : 0;
    }
};

// find ths n-th set bit in mask, starting from base
__device__ __forceinline__ unsigned fns(unsigned mask, unsigned base, int offset)
{
#ifdef __CUDA_ARCH__
    return __fns(mask, base, offset);
#endif
    // unreachable, just for suppress warning
    [[unreachable]] return 0;
}

void Spmv::rbk_spmv(Float                           a,
                    muda::CBCOOMatrixView<Float, 3> A,
                    muda::CDenseVectorView<Float>   x,
                    Float                           b,
                    muda::DenseVectorView<Float>    y)
{
    using namespace muda;
    constexpr int N = 3;
    using T         = Float;

    if(b != 0)
    {
        muda::ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(y.size(),
                   [b = b, y = y.viewer().name("y")] __device__(int i) mutable
                   { y(i) = b * y(i); });
    }
    else
    {
        muda::BufferLaunch().fill<Float>(y.buffer_view(), 0);
    }

    constexpr int          warp_size = 32;
    constexpr unsigned int warp_mask = ~0u;
    constexpr int          block_dim = 128;
    int block_count = (A.triplet_count() + block_dim - 1) / block_dim;

    muda::Launch(block_count, block_dim)
        .kernel_name(__FUNCTION__)
        .apply(
            [a = a,
             A = A.viewer().name("A"),
             x = x.viewer().name("x"),
             b = b,
             y = y.viewer().name("y")] __device__() mutable
            {
                using WarpReduceInt   = hipcub::WarpReduce<int, warp_size>;
                using WarpReduceFloat = hipcub::WarpReduce<Float, warp_size>;
                using WarpScanInt     = hipcub::WarpScan<int>;

                auto global_thread_id   = blockDim.x * blockIdx.x + threadIdx.x;
                auto thread_id_in_block = threadIdx.x;
                auto warp_id            = thread_id_in_block / warp_size;
                auto lane_id            = thread_id_in_block & (warp_size - 1);

                int rest = A.triplet_count() - blockIdx.x * block_dim;
                int valid_count_in_block = rest > block_dim ? block_dim : rest;

                __shared__ union
                {
                    typename WarpReduceInt::TempStorage temp_storage_int[block_dim / warp_size];
                    typename WarpReduceFloat::TempStorage temp_storage_float[block_dim / warp_size];
                };

                int prev_i = -1;
                int next_i = -1;
                int i      = -1;

                Flags   flags;
                Vector3 vec;
                flags.is_cross_warp = 0;


                if(global_thread_id > 0 && global_thread_id < A.triplet_count())
                {
                    auto prev_triplet = A(global_thread_id - 1);
                    prev_i            = prev_triplet.row_index;
                }

                if(global_thread_id < A.triplet_count() - 1)
                {
                    auto next_triplet = A(global_thread_id + 1);
                    next_i            = next_triplet.row_index;
                }

                if(global_thread_id < A.triplet_count())
                {
                    auto Triplet = A(global_thread_id);
                    i            = Triplet.row_index;
                    auto j       = Triplet.col_index;

                    vec = Triplet.value * x.segment<N>(j * N).as_eigen();

                    flags.is_valid = 1;
                }
                else
                {
                    i = -1;
                    vec.setZero();
                    flags.is_valid      = 0;
                    flags.is_cross_warp = 0;
                }

                if(lane_id == 0)
                {
                    flags.is_head = 1;
                    // if this thread is the first thread in the warp
                    // check if the previous triplet is in the same row
                    // if so, this row crosses the warp boundary, we need use atomic add
                    flags.is_cross_warp = b2i(prev_i == i);
                }
                else
                {
                    flags.is_head = b2i(prev_i != i);  // must be 1 or 0, or the result is undefined

                    if(lane_id == warp_size - 1)
                    {
                        // if this thread is the last thread in the warp
                        // check if the next triplet is in the same row
                        // if so, this row crosses the warp boundary, we need use atomic add
                        flags.is_cross_warp = b2i(next_i == i);
                    }
                }

                flags.flags =
                    WarpReduceInt(temp_storage_int[warp_id])
                        .HeadSegmentedReduce(flags.flags, flags.is_head, hipcub::Sum());

                vec.x() = WarpReduceFloat(temp_storage_float[warp_id])
                              .HeadSegmentedReduce(vec.x(), flags.is_head, hipcub::Sum());

                vec.y() = WarpReduceFloat(temp_storage_float[warp_id])
                              .HeadSegmentedReduce(vec.y(), flags.is_head, hipcub::Sum());

                vec.z() = WarpReduceFloat(temp_storage_float[warp_id])
                              .HeadSegmentedReduce(vec.z(), flags.is_head, hipcub::Sum());


                // hipcub::WARP_SYNC(warp_mask);

                flags.is_head = b2i(flags.is_head && flags.is_valid);

                flags.b2i();
                int is_head_mask = hipcub::WARP_BALLOT(flags.is_head, warp_mask);
                uint32_t offset  = fns(is_head_mask, 0, lane_id + 1);

                int valid_bit    = (offset != ~0u);
                int shuffle_mask = hipcub::WARP_BALLOT(valid_bit, warp_mask);

                i = hipcub::ShuffleIndex<32>(i, offset, shuffle_mask);
                flags.flags = hipcub::ShuffleIndex<32>(flags.flags, offset, shuffle_mask);
                vec.x() = hipcub::ShuffleIndex<32>(vec.x(), offset, shuffle_mask);
                vec.y() = hipcub::ShuffleIndex<32>(vec.y(), offset, shuffle_mask);
                vec.z() = hipcub::ShuffleIndex<32>(vec.z(), offset, shuffle_mask);

                if(valid_bit && flags.is_head && flags.is_valid)
                {
                    auto seg_y  = y.segment<N>(i * N);
                    auto result = a * vec;

                    if(flags.is_cross_warp)
                    {
                        seg_y.atomic_add(result.eval());
                    }
                    else
                    {
                        seg_y.as_eigen() += result.eval();
                    }
                }
            });
}

void Spmv::rbk_sym_spmv(Float                           a,
                        muda::CBCOOMatrixView<Float, 3> A,
                        muda::CDenseVectorView<Float>   x,
                        Float                           b,
                        muda::DenseVectorView<Float>    y)

{
    using namespace muda;
    constexpr int N = 3;
    using T         = Float;

    if(b != 0)
    {
        muda::ParallelFor()
            .kernel_name(__FUNCTION__)
            .apply(y.size(),
                   [b = b, y = y.viewer().name("y")] __device__(int i) mutable
                   { y(i) = b * y(i); });
    }
    else
    {
        muda::BufferLaunch().fill<Float>(y.buffer_view(), 0);
    }

    constexpr int warp_size   = 32;
    constexpr int block_dim   = 256;
    int           block_count = (A.triplet_count() + block_dim - 1) / block_dim;

    muda::Launch(block_count, block_dim)
        .file_line(__FILE__, __LINE__)
        .apply(
            [a = a,
             A = A.viewer().name("A"),
             x = x.viewer().name("x"),
             b = b,
             y = y.viewer().name("y")] __device__() mutable
            {
                using WarpReduceInt   = hipcub::WarpReduce<int, warp_size>;
                using WarpReduceFloat = hipcub::WarpReduce<Float, warp_size>;
                using WarpScanInt     = hipcub::WarpScan<int>;

                auto global_thread_id   = blockDim.x * blockIdx.x + threadIdx.x;
                auto thread_id_in_block = threadIdx.x;
                auto warp_id            = thread_id_in_block / warp_size;
                auto lane_id            = thread_id_in_block & (warp_size - 1);

                int rest = A.triplet_count() - blockIdx.x * block_dim;
                int valid_count_in_block = rest > block_dim ? block_dim : rest;

                __shared__ union
                {
                    typename WarpReduceInt::TempStorage temp_storage_int[block_dim / warp_size];
                    typename WarpReduceFloat::TempStorage temp_storage_float[block_dim / warp_size];
                };

                int     prev_i = -1;
                int     next_i = -1;
                int     i      = -1;
                Flags   flags;
                Vector3 vec;

                // In symmtric version, we don't need to check the cross warp
                flags.is_cross_warp = 0;

                // set the previous row index
                if(global_thread_id > 0 && global_thread_id < A.triplet_count())
                {
                    auto prev_triplet = A(global_thread_id - 1);
                    prev_i            = prev_triplet.row_index;
                }

                // set the next row index
                if(global_thread_id < A.triplet_count() - 1 /* && global_thread_id>=0 */)
                {
                    auto next_triplet = A(global_thread_id + 1);
                    next_i            = next_triplet.row_index;
                }

                if(global_thread_id < A.triplet_count())
                {
                    auto Triplet = A(global_thread_id);
                    i            = Triplet.row_index;
                    auto j       = Triplet.col_index;

                    vec = Triplet.value * x.segment<N>(j * N).as_eigen();

                    flags.is_valid = 1;

                    if(i != j)  // process lower triangle
                    {
                        Vector3 vec_ = a * Triplet.value.transpose()
                                       * x.segment<N>(i * N).as_eigen();

                        y.segment<N>(j * N).atomic_add(vec_);
                    }
                }
                else
                {
                    i = -1;
                    vec.setZero();
                    flags.is_valid = 0;
                }

                if(lane_id == 0)
                {
                    flags.is_head = 1;
                }
                else
                {
                    flags.is_head = b2i(prev_i != i);  // must be 1 or 0, or the result is undefined
                }


                // ----------------------------------- warp reduce ----------------------------------------------
                flags.flags =
                    WarpReduceInt(temp_storage_int[warp_id])
                        .HeadSegmentedReduce(flags.flags, flags.is_head, hipcub::Sum());

                vec.x() = WarpReduceFloat(temp_storage_float[warp_id])
                              .HeadSegmentedReduce(vec.x(), flags.is_head, hipcub::Sum());

                vec.y() = WarpReduceFloat(temp_storage_float[warp_id])
                              .HeadSegmentedReduce(vec.y(), flags.is_head, hipcub::Sum());

                vec.z() = WarpReduceFloat(temp_storage_float[warp_id])
                              .HeadSegmentedReduce(vec.z(), flags.is_head, hipcub::Sum());
                // ----------------------------------- warp reduce -----------------------------------------------


                if(flags.is_head && flags.is_valid)
                {
                    auto seg_y  = y.segment<N>(i * N);
                    auto result = a * vec;

                    // Must use atomic add!
                    // Because the same row may be processed by different warps
                    seg_y.atomic_add(result.eval());
                }
            });
}
}  // namespace uipc::backend::cuda
