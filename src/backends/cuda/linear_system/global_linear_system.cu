#include "hip/hip_runtime.h"
#include <linear_system/global_linear_system.h>
#include <linear_system/diag_linear_subsystem.h>
#include <linear_system/off_diag_linear_subsystem.h>
#include <uipc/common/range.h>
#include <linear_system/iterative_solver.h>
#include <linear_system/global_preconditioner.h>
#include <linear_system/local_preconditioner.h>
#include <fstream>

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalLinearSystem);

void GlobalLinearSystem::dump_linear_system(std::string_view filename)
{
    {
        auto& A = m_impl.debug_A;
        m_impl.ctx.convert(m_impl.bcoo_A, A);
        Eigen::MatrixX<Float> mat;
        A.copy_to(mat);

        auto A_file = fmt::format("{}.A.csv", filename);

        std::ofstream file(A_file);
        // dump as .csv file
        for(int i = 0; i < mat.rows(); ++i)
        {
            for(int j = 0; j < mat.cols(); ++j)
            {
                file << mat(i, j) << ",";
            }
            file << "\n";
        }
    }

    {
        Eigen::VectorX<Float> b;
        m_impl.b.copy_to(b);

        auto b_file = fmt::format("{}.b.csv", filename);

        std::ofstream file(b_file);
        // dump as .csv file
        for(int i = 0; i < b.size(); ++i)
        {
            file << b(i) << "\n";
        }
    }

    {
        Eigen::VectorX<Float> x;
        m_impl.x.copy_to(x);

        auto x_file = fmt::format("{}.x.csv", filename);

        std::ofstream file(x_file);
        // dump as .csv file
        for(int i = 0; i < x.size(); ++i)
        {
            file << x(i) << "\n";
        }
    }
}

SizeT GlobalLinearSystem::dof_count() const
{
    return m_impl.b.size();
}

void GlobalLinearSystem::do_build() {}

void GlobalLinearSystem::solve()
{
    m_impl.build_linear_system();
    // if the system is empty, skip the following steps
    if(m_impl.empty_system) [[unlikely]]
        return;
    m_impl.solve_linear_system();
    m_impl.distribute_solution();
}

void GlobalLinearSystem::prepare_hessian()
{
    Timer timer{"Build Linear System"};
    m_impl.empty_system = !m_impl._update_subsystem_extent();
    // if empty, skip the following steps
    if(m_impl.empty_system) [[unlikely]]
        return;

    m_impl._assemble_linear_system();
    m_impl.converter.convert(m_impl.triplet_A, m_impl.bcoo_A);
}

void GlobalLinearSystem::Impl::init()
{
    auto diag_subsystem_view     = diag_subsystems.view();
    auto off_diag_subsystem_view = off_diag_subsystems.view();


    // 1) Record Diag and OffDiag Subsystems
    auto total_count = diag_subsystem_view.size() + off_diag_subsystem_view.size();
    subsystem_infos.resize(total_count);
    // put the diag subsystems in the front
    auto diag_span = span{subsystem_infos}.subspan(0, diag_subsystem_view.size());
    // then the off diag subsystems
    auto off_diag_span = span{subsystem_infos}.subspan(diag_subsystem_view.size(),
                                                       off_diag_subsystem_view.size());
    {
        auto offset = 0;
        for(auto i : range(diag_span.size()))
        {
            auto& dst_diag                  = diag_span[i];
            dst_diag.is_diag                = true;
            dst_diag.local_index            = i;
            auto index                      = offset + i;
            dst_diag.index                  = index;
            diag_subsystem_view[i]->m_index = index;
        }

        offset += diag_subsystem_view.size();
        for(auto i : range(off_diag_span.size()))
        {
            auto& dst_off_diag       = off_diag_span[i];
            dst_off_diag.is_diag     = false;
            dst_off_diag.local_index = i;
            dst_off_diag.index       = offset + i;
        }
    }

    // 2) DoF Offsets/Counts
    accuracy_statisfied_flags.resize(diag_subsystem_view.size());
    {
        diag_dof_offsets_counts.resize(diag_subsystem_view.size());
        auto diag_dof_counts = diag_dof_offsets_counts.counts();
        for(auto&& [i, diag_subsystem] : enumerate(diag_subsystem_view))
        {
            InitDofExtentInfo info;
            diag_subsystem->report_init_extent(info);
            diag_dof_counts[i] = info.m_dof_count;
        }
        diag_dof_offsets_counts.scan();
        auto diag_dof_offsets = diag_dof_offsets_counts.offsets();
        for(auto&& [i, diag_subsystem] : enumerate(diag_subsystem_view))
        {
            InitDofInfo info;
            info.m_dof_offset = diag_dof_offsets[i];
            info.m_dof_count  = diag_dof_counts[i];
            diag_subsystem->receive_init_dof_info(info);
        }
    }

    // 3) Triplet Offsets/Counts
    subsystem_triplet_offsets_counts.resize(total_count);
    off_diag_lr_triplet_counts.resize(off_diag_subsystem_view.size());

    // 4) Preconditioner
    // find out diag systems that don't have preconditioner
    auto local_preconditioner_view = local_preconditioners.view();

    for(auto precond : local_preconditioner_view)
    {
        precond->init();
    }

    for(auto precond : local_preconditioner_view)
    {
        auto index = precond->m_subsystem->m_index;
        diag_span[index].has_local_preconditioner = true;
    }
    no_precond_diag_subsystem_indices.reserve(diag_span.size());
    for(auto&& [i, diag_info] : enumerate(diag_span))
    {
        if(!diag_info.has_local_preconditioner)
        {
            no_precond_diag_subsystem_indices.push_back(i);
        }
    }
}

void GlobalLinearSystem::Impl::build_linear_system()
{
    Timer timer{"Build Linear System"};
    empty_system = !_update_subsystem_extent();
    // if empty, skip the following steps
    if(empty_system) [[unlikely]]
        return;


    _assemble_linear_system();

    converter.convert(triplet_A, bcoo_A);
    converter.ge2sym(bcoo_A);

    _assemble_preconditioner();
}

bool GlobalLinearSystem::Impl::_update_subsystem_extent()
{
    bool dof_count_changed     = false;
    bool triplet_count_changed = false;

    auto diag_subsystem_view       = diag_subsystems.view();
    auto off_diag_subsystem_view   = off_diag_subsystems.view();
    auto diag_dof_counts           = diag_dof_offsets_counts.counts();
    auto diag_dof_offsets          = diag_dof_offsets_counts.offsets();
    auto subsystem_triplet_counts  = subsystem_triplet_offsets_counts.counts();
    auto subsystem_triplet_offsets = subsystem_triplet_offsets_counts.offsets();

    for(const auto& subsystem_info : subsystem_infos)
    {
        if(subsystem_info.is_diag)
        {
            auto           dof_i          = subsystem_info.local_index;
            auto           triplet_i      = subsystem_info.index;
            auto&          diag_subsystem = diag_subsystem_view[dof_i];
            DiagExtentInfo info;
            info.m_storage_type = HessianStorageType::Full;
            diag_subsystem->report_extent(info);

            dof_count_changed |= diag_dof_counts[dof_i] != info.m_dof_count;
            diag_dof_counts[dof_i] = info.m_dof_count;


            triplet_count_changed |= subsystem_triplet_counts[triplet_i] != info.m_block_count;
            subsystem_triplet_counts[triplet_i] = info.m_block_count;
        }
        else
        {
            auto triplet_i = subsystem_info.index;
            auto& off_diag_subsystem = off_diag_subsystem_view[subsystem_info.local_index];
            OffDiagExtentInfo info;
            info.m_storage_type = HessianStorageType::Full;
            off_diag_subsystem->report_extent(info);

            auto total_block_count = info.m_lr_block_count + info.m_rl_block_count;

            triplet_count_changed |= subsystem_triplet_counts[triplet_i] != total_block_count;
            subsystem_triplet_counts[triplet_i] = total_block_count;
            off_diag_lr_triplet_counts[subsystem_info.local_index] =
                ulonglong2{info.m_lr_block_count, info.m_rl_block_count};
        }
    }

    SizeT total_dof     = 0;
    SizeT total_triplet = 0;

    if(dof_count_changed)
    {
        diag_dof_offsets_counts.scan();
    }
    total_dof = diag_dof_offsets_counts.total_count();
    if(x.capacity() < total_dof)
    {
        auto reserve_count = total_dof * reserve_ratio;
        x.reserve(reserve_count);
        b.reserve(reserve_count);
    }
    auto blocked_dof = total_dof / DoFBlockSize;
    triplet_A.reshape(blocked_dof, blocked_dof);
    x.resize(total_dof);
    b.resize(total_dof);

    if(triplet_count_changed) [[likely]]
    {
        subsystem_triplet_offsets_counts.scan();
    }
    total_triplet = subsystem_triplet_offsets_counts.total_count();

    if(triplet_A.triplet_capacity() < total_triplet)
    {
        auto reserve_count = total_triplet * reserve_ratio;
        triplet_A.reserve_triplets(reserve_count);
        bcoo_A.reserve_triplets(reserve_count);
    }
    triplet_A.resize_triplets(total_triplet);

    if(total_dof == 0 || total_triplet == 0) [[unlikely]]
    {
        spdlog::warn("The global linear system is empty, skip *assembling, *solving and *solution distributing phase.");
        return false;
    }

    return true;
}

void GlobalLinearSystem::Impl::_assemble_linear_system()
{
    auto HA = triplet_A.view();
    auto B  = b.view();

    auto diag_subsystem_view     = diag_subsystems.view();
    auto off_diag_subsystem_view = off_diag_subsystems.view();

    auto diag_dof_counts  = diag_dof_offsets_counts.counts();
    auto diag_dof_offsets = diag_dof_offsets_counts.offsets();

    auto subsystem_triplet_counts  = subsystem_triplet_offsets_counts.counts();
    auto subsystem_triplet_offsets = subsystem_triplet_offsets_counts.offsets();

    for(const auto& subsystem_info : subsystem_infos)
    {
        if(subsystem_info.is_diag)
        {
            auto  dof_i          = subsystem_info.local_index;
            auto  triplet_i      = subsystem_info.index;
            auto& diag_subsystem = diag_subsystem_view[dof_i];

            int  dof_offset         = diag_dof_offsets[dof_i];
            int  dof_count          = diag_dof_counts[dof_i];
            int  blocked_dof_offset = dof_offset / DoFBlockSize;
            int  blocked_dof_count  = dof_count / DoFBlockSize;
            int2 ij_offset          = {blocked_dof_offset, blocked_dof_offset};
            int2 ij_count           = {blocked_dof_count, blocked_dof_count};

            DiagInfo info{this};

            info.m_index        = triplet_i;
            info.m_storage_type = HessianStorageType::Full;
            info.m_gradient     = B.subview(dof_offset, dof_count);
            info.m_hessian = HA.subview(subsystem_triplet_offsets[triplet_i],
                                        subsystem_triplet_counts[triplet_i])
                                 .submatrix(ij_offset, ij_count);

            diag_subsystem->assemble(info);
        }
        else
        {
            auto triplet_i   = subsystem_info.index;
            auto local_index = subsystem_info.local_index;
            auto& off_diag_subsystem = off_diag_subsystem_view[subsystem_info.local_index];
            auto& l_diag_index = off_diag_subsystem->m_l->m_index;
            auto& r_diag_index = off_diag_subsystem->m_r->m_index;


            int l_blocked_dof_offset = diag_dof_offsets[l_diag_index] / DoFBlockSize;
            int l_blocked_dof_count = diag_dof_counts[l_diag_index] / DoFBlockSize;

            int r_blocked_dof_offset = diag_dof_offsets[r_diag_index] / DoFBlockSize;
            int r_blocked_dof_count = diag_dof_counts[r_diag_index] / DoFBlockSize;

            auto lr_triplet_offset = subsystem_triplet_offsets[triplet_i];
            auto lr_triplet_count  = off_diag_lr_triplet_counts[local_index].x;
            auto rl_triplet_offset = lr_triplet_offset + lr_triplet_count;
            auto rl_triplet_count  = off_diag_lr_triplet_counts[local_index].y;

            OffDiagInfo info{this};
            info.m_index        = triplet_i;
            info.m_storage_type = HessianStorageType::Full;

            info.m_lr_hessian =
                HA.subview(lr_triplet_offset, lr_triplet_count)
                    .submatrix(int2{l_blocked_dof_offset, r_blocked_dof_offset},
                               int2{l_blocked_dof_count, r_blocked_dof_count});

            info.m_rl_hessian =
                HA.subview(rl_triplet_offset, rl_triplet_count)
                    .submatrix(int2{r_blocked_dof_offset, l_blocked_dof_offset},
                               int2{r_blocked_dof_count, l_blocked_dof_count});

            // spdlog::info("rl_offset: {}, lr_offset: {}", rl_triplet_offset, lr_triplet_offset);

            off_diag_subsystem->assemble(info);
        }
    }
}

void GlobalLinearSystem::Impl::_assemble_preconditioner()
{
    if(global_preconditioner)
    {
        GlobalPreconditionerAssemblyInfo info{this};
        global_preconditioner->assemble(info);
    }

    for(auto&& preconditioner : local_preconditioners.view())
    {
        LocalPreconditionerAssemblyInfo info{this, preconditioner->m_subsystem->m_index};
        preconditioner->assemble(info);
    }
}

void GlobalLinearSystem::Impl::solve_linear_system()
{
    Timer timer{"Solve Linear System"};
    if(iterative_solver)
    {
        SolvingInfo info{this};
        info.m_b = b.cview();
        info.m_x = x.view();
        iterative_solver->solve(info);
        spdlog::info("Iterative linear solver iteration count: {}", info.m_iter_count);
    }
}

void GlobalLinearSystem::Impl::distribute_solution()
{
    auto diag_subsystem_view = diag_subsystems.view();
    auto diag_dof_counts     = diag_dof_offsets_counts.counts();
    auto diag_dof_offsets    = diag_dof_offsets_counts.offsets();

    // distribute the solution to all diag subsystems
    for(auto&& [i, diag_subsystem] : enumerate(diag_subsystems.view()))
    {
        SolutionInfo info{this};
        info.m_solution = x.view().subview(diag_dof_offsets[i], diag_dof_counts[i]);
        diag_subsystem->retrieve_solution(info);
    }
}

void GlobalLinearSystem::Impl::apply_preconditioner(muda::DenseVectorView<Float> z,
                                                    muda::CDenseVectorView<Float> r)
{
    auto diag_dof_counts  = diag_dof_offsets_counts.counts();
    auto diag_dof_offsets = diag_dof_offsets_counts.offsets();

    if(global_preconditioner)
    {
        ApplyPreconditionerInfo info{this};
        info.m_z = z;
        info.m_r = r;
        global_preconditioner->apply(info);
    }

    for(auto& preconditioner : local_preconditioners.view())
    {
        ApplyPreconditionerInfo info{this};
        auto                    index  = preconditioner->m_subsystem->m_index;
        auto                    offset = diag_dof_offsets[index];
        auto                    count  = diag_dof_counts[index];
        info.m_z                       = z.subview(offset, count);
        info.m_r                       = r.subview(offset, count);
        preconditioner->apply(info);
    }

    if(!global_preconditioner)
    {
        for(auto i : no_precond_diag_subsystem_indices)
        {
            auto offset = diag_dof_offsets[i];
            auto count  = diag_dof_counts[i];
            auto z_sub  = z.subview(offset, count);
            auto r_sub  = r.subview(offset, count);
            z_sub.buffer_view().copy_from(r_sub.buffer_view());
        }
    }
}

void GlobalLinearSystem::Impl::spmv(Float                         a,
                                    muda::CDenseVectorView<Float> x,
                                    Float                         b,
                                    muda::DenseVectorView<Float>  y)
{
    spmver.rbk_sym_spmv(a, bcoo_A.cview(), x, b, y);
}

bool GlobalLinearSystem::Impl::accuracy_statisfied(muda::DenseVectorView<Float> r)
{
    auto diag_dof_counts  = diag_dof_offsets_counts.counts();
    auto diag_dof_offsets = diag_dof_offsets_counts.offsets();

    for(auto&& [i, diag_subsystems] : enumerate(diag_subsystems.view()))
    {
        AccuracyInfo info{this};
        info.m_r = r.subview(diag_dof_offsets[i], diag_dof_counts[i]);
        diag_subsystems->accuracy_check(info);

        accuracy_statisfied_flags[i] = info.m_statisfied ? 1 : 0;
    }

    return std::ranges::all_of(accuracy_statisfied_flags,
                               [](bool flag) { return flag; });
}

void GlobalLinearSystem::DiagExtentInfo::extent(SizeT hessian_block_count, SizeT dof_count) noexcept
{
    m_block_count = hessian_block_count;
    UIPC_ASSERT(dof_count % DoFBlockSize == 0,
                "dof_count must be multiple of {}, yours {}.",
                DoFBlockSize,
                dof_count);
    m_dof_count = dof_count;
}

void GlobalLinearSystem::OffDiagExtentInfo::extent(SizeT lr_hessian_block_count,
                                                   SizeT rl_hassian_block_count) noexcept
{
    m_lr_block_count = lr_hessian_block_count;
    m_rl_block_count = rl_hassian_block_count;
}
auto GlobalLinearSystem::AssemblyInfo::A() const -> CBCOOMatrixView
{
    return m_impl->bcoo_A.cview();
}

auto GlobalLinearSystem::AssemblyInfo::storage_type() const -> HessianStorageType
{
    return HessianStorageType::Symmetric;
}
SizeT GlobalLinearSystem::LocalPreconditionerAssemblyInfo::dof_offset() const
{
    auto diag_dof_offsets = m_impl->diag_dof_offsets_counts.offsets();
    return diag_dof_offsets[m_index];
}
SizeT GlobalLinearSystem::LocalPreconditionerAssemblyInfo::dof_count() const
{
    auto diag_dof_counts = m_impl->diag_dof_offsets_counts.counts();
    return diag_dof_counts[m_index];
}
}  // namespace uipc::backend::cuda

namespace uipc::backend::cuda
{
void GlobalLinearSystem::add_subsystem(DiagLinearSubsystem* subsystem)
{
    check_state(SimEngineState::BuildSystems, "add_subsystem()");
    UIPC_ASSERT(subsystem != nullptr, "The subsystem should not be nullptr.");
    m_impl.diag_subsystems.register_subsystem(*subsystem);
}

void GlobalLinearSystem::add_subsystem(OffDiagLinearSubsystem* subsystem)
{
    check_state(SimEngineState::BuildSystems, "add_subsystem()");
    m_impl.off_diag_subsystems.register_subsystem(*subsystem);
}

void GlobalLinearSystem::add_solver(IterativeSolver* solver)
{
    check_state(SimEngineState::BuildSystems, "add_solver()");
    UIPC_ASSERT(solver != nullptr, "The solver should not be nullptr.");
    m_impl.iterative_solver.register_subsystem(*solver);
}

void GlobalLinearSystem::add_preconditioner(LocalPreconditioner* preconditioner)
{
    check_state(SimEngineState::BuildSystems, "add_preconditioner()");
    UIPC_ASSERT(preconditioner != nullptr, "The preconditioner should not be nullptr.");
    m_impl.local_preconditioners.register_subsystem(*preconditioner);
}

void GlobalLinearSystem::add_preconditioner(GlobalPreconditioner* preconditioner)
{
    check_state(SimEngineState::BuildSystems, "add_preconditioner()");
    UIPC_ASSERT(preconditioner != nullptr, "The preconditioner should not be nullptr.");
    m_impl.global_preconditioner.register_subsystem(*preconditioner);
}

void GlobalLinearSystem::init()
{
    m_impl.init();
}
}  // namespace uipc::backend::cuda