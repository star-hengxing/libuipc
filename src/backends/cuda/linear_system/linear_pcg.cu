#include "hip/hip_runtime.h"
#include <linear_system/linear_pcg.h>
#include <sim_engine.h>
#include <linear_system/global_linear_system.h>
namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(LinearPCG);

void LinearPCG::do_build(BuildInfo& info)
{
    auto& global_linear_system = require<GlobalLinearSystem>();

    // TODO: get info from the scene, now we just use the default value
    max_iter_ratio  = 2;
    global_tol_rate = world().scene().info()["linear_system"]["tol_rate"];
    // spdlog::info("LinearPCG: max_iter_ratio = {}, tol_rate = {}", max_iter_ratio, global_tol_rate);
}

void LinearPCG::do_solve(GlobalLinearSystem::SolvingInfo& info)
{
    auto x = info.x();
    auto b = info.b();

    x.buffer_view().fill(0);

    auto N = x.size();
    if(z.capacity() < N)
    {
        auto M = reserve_ratio * N;
        z.reserve(M);
        p.reserve(M);
        r.reserve(M);
        Ap.reserve(M);
    }

    z.resize(N);
    p.resize(N);
    r.resize(N);
    Ap.resize(N);

    auto iter = pcg(x, b, max_iter_ratio * b.size());

    info.iter_count(iter);
}

SizeT LinearPCG::pcg(muda::DenseVectorView<Float> x, muda::CDenseVectorView<Float> b, SizeT max_iter)
{
    SizeT k = 0;
    // r = b - A * x
    {
        // r = b;
        r.buffer_view().copy_from(b.buffer_view());

        // x == 0, so we don't need to do the following
        // r = - A * x + r
        //spmv(-1.0, x.as_const(), 1.0, r.view());
    }

    Float alpha, beta, rz, rz0;

    // z = P * r (apply preconditioner)
    apply_preconditioner(z, r);

    // p = z
    p = z;

    // init rz
    // rz = r^T * z
    rz = ctx().dot(r.cview(), z.cview());

    rz0 = std::abs(rz);

    if constexpr(RUNTIME_CHECK)
    {
        if(std::isnan(rz0) || !std::isfinite(rz0))
        {
            auto norm_r = ctx().norm(r.cview());
            auto norm_z = ctx().norm(z.cview());

            UIPC_ASSERT(!std::isnan(rz0) && std::isfinite(rz0),
                        "Init Residual is {}, norm(r) = {}, norm(z) = {}",
                        rz0,
                        norm_r,
                        norm_z);
        }
    }

    // check convergence
    if(accuracy_statisfied(r) && rz0 == Float{0.0})
        return 0;

    for(k = 1; k < max_iter; ++k)
    {
        spmv(p.cview(), Ap.view());

        // alpha = rz / dot(p.cview(), Ap.cview());
        alpha = rz / ctx().dot(p.cview(), Ap.cview());

        // x = x + alpha * p
        ctx().axpby(alpha, p.cview(), Float{1}, x);

        // r = r - alpha * Ap
        ctx().axpby(-alpha, Ap.cview(), Float{1}, r.view());

        // z = P * r (apply preconditioner)
        apply_preconditioner(z, r);

        // rz_new = r^T * z
        // rz_new = dot(r.cview(), z.cview());
        Float rz_new = ctx().dot(r.cview(), z.cview());

        if constexpr(RUNTIME_CHECK)
        {
            if(std::isnan(rz_new) || !std::isfinite(rz_new))
            {
                auto norm_r = ctx().norm(r.cview());
                auto norm_z = ctx().norm(z.cview());
                UIPC_ASSERT(!std::isnan(rz_new) && std::isfinite(rz_new),
                            "Residual is {}, norm(r) = {}, norm(z) = {}",
                            rz_new,
                            norm_r,
                            norm_z);
            }
        }

        // check convergence
        if(accuracy_statisfied(r) && std::abs(rz_new) <= global_tol_rate * rz0)
            break;

        // beta = rz_new / rz
        beta = rz_new / rz;

        // p = z + beta * p
        ctx().axpby(Float{1}, z.cview(), beta, p.view());

        // update rz
        rz = rz_new;
    }

    return k;
}
}  // namespace uipc::backend::cuda
